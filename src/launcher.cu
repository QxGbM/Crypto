

#include <definitions.cuh>
#include <launcher.cuh>

#include <timer.cuh>
#include <h_ops/dev_hierarchical_index.cuh>
#include <h_ops/dev_hierarchical_ops_dag.cuh>
#include <h_ops/dev_hierarchical_ops_tree.cuh>
#include <h_ops/dev_hierarchical_ops.cuh>
#include <dev_temp.cuh>
#include <matrix/dev_hierarchical.cuh>
#include <instructions/instructions_scheduler.cuh>
#include <instructions/instructions_manager.cuh>
#include <kernel.cuh>

void print_dev_mat (real_t * dev_mat, const int nx, const int ny)
{
   real_t * data = new real_t [(size_t) nx * ny];
   hipMemcpy (data, dev_mat, (size_t) nx * ny * sizeof(real_t), hipMemcpyDeviceToHost);
   for (int i = 0; i < ny; i++)
   {
     for (int j = 0; j < nx; j++)
     { printf("%e ", data[i * nx + j]); }
     printf("\n");
   }
   delete[] data;
}

hipError_t allocate_clocks (unsigned long long *** clocks, const int workers, const int * lengths)
{
  unsigned long long ** tmp = new unsigned long long * [workers];
  hipMalloc(clocks, workers * sizeof(unsigned long long *));

  for (int i = 0; i < workers; i++)
  {
    hipMalloc(&tmp[i], ((size_t) 1 + lengths[i]) * sizeof(unsigned long long));
    hipMemset(tmp[i], 0, ((size_t)1 + lengths[i]) * sizeof(unsigned long long));
  }
  hipMemcpy(* clocks, tmp, workers * sizeof(unsigned long long *), hipMemcpyHostToDevice);

  return hipGetLastError();
}

hipError_t generateLaunchArgsFromTree (int *** dev_insts, void *** dev_ptrs, int ** comm_space, real_t *** block_tmps, real_t ** dev_rnd_seed, unsigned long long *** clocks,
  instructions_scheduler ** schedule_addr, double * total_lapse, long long * flops, const h_ops_tree * tree, real_t ** tmp_ptrs, const int workers, const int start_index, const int length_max)
{
  double clock_start, clock_end, clock_lapse, clock_total = 0.;
  printf("-- Host Summary: -- \n");

  clock_start = omp_get_wtime();
  h_ops_dag dag = h_ops_dag (tree, start_index, length_max);
  clock_end = omp_get_wtime();
  clock_lapse = clock_end - clock_start;
  clock_total += clock_lapse;
  printf("DAG Created in %f ms.\n", 1000. * clock_lapse); dag.print(); std::cout << "Density: " << dag.dag_density() << std::endl;

  clock_start = omp_get_wtime();
  * schedule_addr = new instructions_scheduler (&dag, workers);
  clock_end = omp_get_wtime();
  clock_lapse = clock_end - clock_start;
  clock_total += clock_lapse;
  printf("Schedule Created in %f ms.\n", 1000. * clock_lapse); //schedule.print();

  int * lengths = (* schedule_addr) -> getLengths();
  allocate_clocks(clocks, workers, lengths);
  delete lengths;

  clock_start = omp_get_wtime();
  instructions_manager ins = instructions_manager (workers, &dag, * schedule_addr, (void **) tmp_ptrs);
  clock_end = omp_get_wtime();
  clock_lapse = clock_end - clock_start;
  clock_total += clock_lapse;
  printf("Instruction generated in %f ms.\n", 1000. * clock_lapse); //ins.print();

  clock_start = omp_get_wtime();
  hipError_t error = ins.getLaunchArgs(dev_insts, dev_ptrs, comm_space, block_tmps, dev_rnd_seed, _SEED);
  clock_end = omp_get_wtime();
  clock_lapse = clock_end - clock_start;
  clock_total += clock_lapse;
  printf("Args generated in %f ms.\n", 1000. * clock_lapse);
  fprintf(stderr, "-- Host Args Generation: %s. --\n\n", hipGetErrorString(error));

  * total_lapse = clock_total;
  * flops = dag.getFlops();
  return error;
}

hipError_t launchKernelWithArgs (int ** dev_insts, void ** dev_ptrs, int * comm_space, real_t ** block_tmps, real_t * dev_rnd_seed, unsigned long long ** clocks, 
  const int workers, const int num_threads, hipStream_t main_stream)
{
  void ** args = new void * [6] { &dev_insts, &dev_ptrs, &comm_space, &block_tmps, &dev_rnd_seed, &clocks };
  hipError_t error = hipLaunchKernel((void *) kernel_dynamic, workers, num_threads, args, 0, main_stream);
  fprintf(stderr, "Kernel Launch: %s\n\n", hipGetErrorString(error));

  /*hipDeviceSynchronize();
  for (int i = 0; i < workers; i++)
  {
    hipFree(dev_insts[i]); // creates seg fault due to dev_insts is on device;
    if (block_tmps[i] != nullptr)
    { hipFree(block_tmps[i]); }
  }*/

  hipFree(dev_insts);
  hipFree(dev_ptrs);
  hipFree(comm_space);
  hipFree(block_tmps);
  hipFree(dev_rnd_seed);
  delete[] args;

  return error;
}

hipError_t hierarchical_GETRF (dev_hierarchical * h, const int num_blocks, const int num_threads, const int kernel_size)
{
  hipSetDevice(0);
  if (sizeof(real_t) == 8 && hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) == hipSuccess)
  { printf("Shared memory bank size configured to be 8-bytes.\n"); }

  hipDeviceProp_t deviceprop;
  hipGetDeviceProperties(&deviceprop, 0);
  int numSMs = deviceprop.multiProcessorCount, numBlocksPerSm = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void *) kernel_dynamic, num_threads, 0);
  printf("# SMs: %d, # Blocks per SM for launch: %d\n\n", numSMs, numBlocksPerSm);

  const int workers_max = numSMs * numBlocksPerSm, workers = workers_max < num_blocks ? workers_max : num_blocks;
  if (workers == 0)
  { printf("Launch Config: Too many resources requested for launch.\n\n"); return hipErrorInvalidConfiguration; }
  else if (workers < num_blocks)
  { printf("Launch Config: Number of launched blocks reduced from %d to %d. \n\n", num_blocks, workers); }

  const int ny = h -> getNy_abs(), nx = h -> getNx_abs();
  printf("Start Testing Hierarchical - LU for: %d x %d.\n\n", ny, nx);

  timer myTimer = timer();
  hipStream_t main_stream;
  hipStreamCreate(&main_stream);

  double clock_start, clock_end, clock_lapse;
  hipError_t error = hipSuccess;
  dev_temp tmp_mngr = dev_temp();

  clock_start = omp_get_wtime();
  const h_index * root = h -> getRootIndex();
  const h_ops_tree * tree = h -> generateOps_GETRF(root, &tmp_mngr);
  clock_end = omp_get_wtime();
  clock_lapse = clock_end - clock_start;
  printf("Tree Generated in %f ms.\n\n", 1000. * clock_lapse); //tree->print();
  delete root;

  real_t ** tmp_ptrs = tmp_mngr.allocate(), ** block_tmps, * dev_rnd_seed;
  int ** dev_insts, * comm_space, iters = kernel_size <= 0 ? 1 : (tree -> length() + kernel_size - 1) / kernel_size;
  void ** dev_ptrs;
  long long int exeFLOPS = 0, tmp;
  unsigned long long int ** clocks;
  char event_name[32];

  for (int i = 0; i < iters && error == hipSuccess; i++)
  {
    instructions_scheduler * schedule;
    error = generateLaunchArgsFromTree (&dev_insts, &dev_ptrs, &comm_space, &block_tmps, &dev_rnd_seed, &clocks, &schedule, &clock_lapse, &tmp, tree, tmp_ptrs, workers, i * kernel_size, kernel_size);
    printf("Host %f ms.\n\n", 1000. * clock_lapse);
    exeFLOPS += tmp;

    sprintf(event_name, "Kernel %d", i);

    myTimer.newEvent(event_name, start, main_stream);
    error = launchKernelWithArgs (dev_insts, dev_ptrs, comm_space, block_tmps, dev_rnd_seed, clocks, workers, num_threads, main_stream);
    myTimer.newEvent(event_name, end, main_stream);

    //schedule -> analyzeClocks(clocks);
    delete schedule;
  }

  const double exeTime = myTimer.dumpAllEvents_Sync();

  hipFree(tmp_ptrs[0]);
  delete[] tmp_ptrs;

  const long long int estFLOPS = h_ops::getFlops_GETRF(&tmp, nx, ny);
  const double compressRatio = estFLOPS == 0 ? 0 : 100. * exeFLOPS / estFLOPS;

  printf("-- Kernel Running Summary --\n"
    "Actual FLOPS: %llu.\nDense-LU FLOPS: %llu.\nFLOPS Compression Ratio: %f%%.\n", 
    exeFLOPS, estFLOPS, compressRatio);

  double gpuflops = 1.e3 * exeFLOPS / exeTime;
  int power = 0;

  while (power < 4 && gpuflops > 1.e3) 
  { gpuflops *= 1.e-3; power ++; }
  printf("GPU: %f ", gpuflops);

  switch (power)
  {
  case 0: break;
  case 1: printf("K"); break;
  case 2: printf("M"); break;
  case 3: printf("G"); break;
  case 4: printf("T"); break;
  }
  printf("FLOPS/S.\n");

  gpuflops *= compressRatio == 0 ? 0 : 100. / compressRatio;

  while (power < 4 && gpuflops > 1.e3) 
  { gpuflops *= 1.e-3; power ++; }
  printf("Equivalent Dense-LU: %f ", gpuflops);

  switch (power)
  {
  case 0: break;
  case 1: printf("K"); break;
  case 2: printf("M"); break;
  case 3: printf("G"); break;
  case 4: printf("T"); break;
  }
  printf("FLOPS/S.\n\n");

  error = hipStreamDestroy(main_stream);

  return error;
}

