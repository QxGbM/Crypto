
#include <kernel_cublas.cuh>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsolver.h>

using std::vector;

void kernel_cublas(const int n_streams, const int n_insts, const vector<int>* insts, vector<double*> ptrs) {

  hipEvent_t* comm = new hipEvent_t[n_insts];
  for (int i = 0; i < n_insts; i++)
  { hipEventCreate(&comm[i]); }

#pragma omp parallel for
  for (int i = 0; i < n_streams; i++) {
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipsolverHandle_t solvHandle;
    hipblasHandle_t blasHandle;

    hipsolverDnCreate(&solvHandle);
    hipsolverSetStream(solvHandle, stream);

    double* workspace;
    hipMalloc(&workspace, 16384);

    hipblasCreate(&blasHandle);
    hipblasSetStream(blasHandle, stream);
    hipblasSetWorkspace(blasHandle, workspace, 16384);

    const int* pc = insts[i].data();
    int next_pc = 0;

  load:
    pc = &pc[next_pc];
    switch ((opcode_t)pc[0]) {
    case execute:
    { goto exe; }
    case signal_wait:
    { goto wait; }
    case finish: default:
    { goto fin; }
    }

  exe:
    switch ((operation_t)pc[2]) {
    case nop:
    { next_pc = nop_l; goto load; }
    case getrf: 
    {
      double* A = ptrs[pc[3]];
      int offset = pc[4], m = pc[5], n = pc[6], ld = pc[7];
      hipsolverDnDgetrf(solvHandle, m, n, &A[offset], ld, workspace, nullptr, nullptr);
      hipEventRecord(comm[pc[1]], stream);
      next_pc = 8;
      goto load;
    }
    case trsml:
    {
      double* B = ptrs[pc[3]], * L = ptrs[pc[4]];
      int offset_b = pc[5], offset_l = pc[6], n = pc[7], m = pc[8], ld_b = pc[10], ld_l = pc[11];
      bool b_T = (bool) pc[12];
      const double alpha = 1;
      hipblasDtrsm(blasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, m, n, &alpha, &L[offset_l], ld_l, &B[offset_b], ld_b);
      hipEventRecord(comm[pc[1]], stream);
      next_pc = 13;
      goto load;
    }
    /*case trsmr:
    {
      double* B = ptrs[pc[3]], * U = ptrs[pc[4]];
      int offset_b = pc[5], offset_u = pc[6], n = pc[7], m = pc[8], ld_b = pc[10], ld_u = pc[11];
      bool b_T = (bool) pc[12];
      const double alpha = 1;
      if (b_T)
      { hipblasDtrsm(blasHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_FULL, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha, &U[offset_u], ld_u, &B[offset_b], ld_b); }
      else
      { hipblasDtrsm(blasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_FULL, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha, &U[offset_u], ld_u, &B[offset_b], ld_b); }
      hipEventRecord(comm[pc[1]], stream);
      next_pc = 13; 
      goto load;
    }
    case gemm:
    {
      double* M = ptrs[pc[3]], * A = ptrs[pc[4]], * B = ptrs[pc[5]];
      int offset_m = pc[6], offset_a = pc[7], offset_b = pc[8], m = pc[9], n = pc[10], k = pc[11], ld_m = pc[12], ld_a = pc[13], ld_b = pc[14];
      bool a_T = (bool)pc[15], b_T = (bool)pc[16];
      double alpha = -1, beta = 1;
      hipblasDgemm(blasHandle, (hipblasOperation_t)a_T, (hipblasOperation_t)b_T, m, n, k, &alpha, &A[offset_a], ld_a, &B[offset_b], ld_b, &beta, &M[offset_m], ld_m);
      hipEventRecord(comm[pc[1]], stream);
      next_pc = 17; 
      goto load;
    }*/
    default:
      goto fin;

    }

  wait:
    hipEventSynchronize(comm[pc[1]]);
    next_pc = 3;

  fin:
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    hipFree(workspace);
    hipblasDestroy(blasHandle);
    hipsolverDnDestroy(solvHandle);

  }

  hipDeviceSynchronize();

  for (int i = 0; i < n_insts; i++)
  { hipEventDestroy(comm[i]); }
  delete[] comm;

  return;
}



