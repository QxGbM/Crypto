#include "hip/hip_runtime.h"

#include <matrix/LowRank.cuh>

LowRank::LowRank(const int m, const int n, const int rank, const int ld) : Element (element_t::low_rank, 0, 0) {
  U = new Clusterbasis(m, rank, &(Element::abs_y), ld);
  V = new Clusterbasis(n, rank, &(Element::abs_x), ld);
  S = new Dense(rank, rank);
}

LowRank::LowRank(const int m, const int n, const int rank, const int abs_y, const int abs_x, const int ld) : Element(element_t::low_rank, abs_y, abs_x) {
  U = new Clusterbasis(m, rank, &(Element::abs_y), ld);
  V = new Clusterbasis(n, rank, &(Element::abs_x), ld);
  S = new Dense(rank, rank);
}

LowRank::~LowRank() {
  delete U;
  delete V;
  if (S != nullptr)
  { delete S; }
  Element::~Element();
}

LowRank* LowRank::getElementLowRank()
{ return this; }

int LowRank::getRowDimension() const 
{ return U->getDimension(); }

int LowRank::getColumnDimension() const
{ return V->getDimension(); }

int LowRank::getRank () const {
  using std::min;
  return min(U->getRank(), V->getRank()); 
}

Dense* LowRank::convertToDense() const {
  return nullptr;
}

void LowRank::print() const {
  
}

void LowRank::print(vector<int>& indices, vector<int>& config) const {

}

Clusterbasis* LowRank::getU() {
  return U;
}

Clusterbasis* LowRank::getV() {
  return V;
}


/*
h_ops_tree * LowRank::generateOps_GETRF (const h_index * self, dev_temp * tmp_mngr)
{ 
  printf("Error: GETRF should not be performed on low-rank matrices.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSML (const h_index * self, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSML should not have a low-rank matrix be the lower triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSML (const h_index * self, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSML should not have a low-rank matrix be the lower triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSML (const h_index * self, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSM should not have a low-rank matrix be the lower triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSML (const h_index * self, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSM should not have a low-rank matrix be the lower triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSMR (const h_index * self, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSMR should not have a low-rank matrix be the upper triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSMR (const h_index * self, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSMR should not have a low-rank matrix be the upper triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSMR (const h_index * self, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSMR should not have a low-rank matrix be the upper triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_TRSMR (const h_index * self, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  printf("Error: TRSMR should not have a low-rank matrix be the upper triangular.\n");
  return nullptr;
}

h_ops_tree * LowRank::generateOps_ACCM (const h_index * self, const h_index * index_tmp_lr)
{
  return new h_ops_tree (accum, self, index_tmp_lr); 
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Dense * A, const h_index * index_a, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  if (!self -> isLowRank_Full())
  {
    printf("WARNING: Potential Accuracy Loss from an attempt to accumulate Dense into Low-Rank.\n");
    op -> resizeChildren (2);

    int block_id, tmp_size = self -> getSize();
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self); index_tmp.setTemp_Dense (block_id);

    h_ops_tree * op_ = new h_ops_tree (gemm, &index_tmp, index_a, index_b);
    op -> setChild (op_, 0);
    delete op_;

    op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, 1);
    delete op_;
  }

  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const LowRank * A, const h_index * index_a, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b), * op_;

  if (!(self -> isVT() && index_a -> isVT()))
  {
    op -> resizeChildren (2);

    int rank_a = index_a -> getRank(), tmp_size = rank_a * index_b -> getNx(self -> getNx()), block_id;
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self), index_av = h_index (index_a);
    index_tmp.setTemp_Low_Rank (block_id, rank_a);
    index_tmp.setU_data (index_a);

    op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild(op_, 1);
    delete op_;

    index_tmp.setVT();
    index_av.setVT();

    op_ = new h_ops_tree (gemm, &index_tmp, &index_av, index_b);
    op -> setChild (op_, 0);
    delete op_;
  }

  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Hierarchical * A, const h_index * index_a, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  const int n_k = A -> getNx_blocks(), n_mk = n_k * A -> getNy_blocks();
  int * y, * k, x = self -> getNx(index_b -> getNx());
  A -> getOffsets_y(&y);
  A -> getOffsets_x(&k);

  if (self -> isLowRank_Full())
  {
    printf("WARNING: Potential Accuracy Loss from an attempt to accumulate Dense into Low-Rank.\n");

    op -> resizeChildren(n_mk + 1);

    int block_id, tmp_size = self -> getSize();
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self); index_tmp.setTemp_Dense(block_id);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, n_mk);
    delete op_;

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mk; i++)
    {
      const int row = i / n_k, col = i - row * n_k;
      const h_index index_ai = h_index (A, index_a, row, col), index_m = h_index (&index_tmp, y[row], 0, index_ai.getNy(), x), index_bj = h_index (index_b, k[col], 0, index_ai.getNx(), x);
      h_ops_tree * op_i = Dense::generateOps_GEMM(&index_m, A -> getElement_blocks(row, col), &index_ai, B, &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }
  else
  {
    op -> resizeChildren(n_mk);

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mk; i++)
    {
      const int row = i / n_k, col = i - row * n_k;
      const h_index index_ai = h_index (A, index_a, row, col), index_m = h_index (self, y[row], 0, index_ai.getNy(), x), index_bj = h_index (index_b, k[col], 0, index_ai.getNx(), x);
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A -> getElement_blocks(row, col), &index_ai, B, &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }

  delete[] y;
  delete[] k;
  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Element * A, const h_index * index_a, const Dense * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_a = A -> getElementHierarchical();
  const LowRank *lr_a = A -> getElementLowRank();
  const Dense *d_a = A -> getElementDense();

  if (d_a != nullptr)
  { return generateOps_GEMM (self, d_a, index_a, B, index_b, tmp_mngr); }
  if (lr_a != nullptr)
  { return generateOps_GEMM (self, lr_a, index_a, B, index_b, tmp_mngr); }
  if (h_a != nullptr)
  { return generateOps_GEMM (self, h_a, index_a, B, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Dense * A, const h_index * index_a, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  if (!(self -> isU() && index_b -> isU()))
  {
    op -> resizeChildren (2);
    int rank_b = index_b -> getRank(), tmp_size = rank_b * index_a -> getNy(self -> getNy()), block_id;
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self), index_bu = h_index (index_b);
    index_tmp.setTemp_Low_Rank (block_id, rank_b);
    index_tmp.setVT_data (index_b);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, 1);
    delete op_;

    index_tmp.setU();
    index_bu.setU();

    op_ = new h_ops_tree (gemm, &index_tmp, index_a, &index_bu);
    op -> setChild (op_, 0);
    delete op_;
  }

  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const LowRank * A, const h_index * index_a, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  if (self -> isLowRank_Full())
  {
    op -> resizeChildren (2);

    h_index index_tmp = h_index (self);
    bool a; 
    int rank = index_a -> getMinRank (index_b, &a);
    int tmp_size = rank * (a ? index_b -> getNx(self -> getNx()) : index_a -> getNy(self -> getNy()));
    int block_id;

#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    index_tmp.setTemp_Low_Rank(block_id, rank);
    if (a)
    { index_tmp.setU_data(index_a); }
    else
    { index_tmp.setVT_data(index_b); }

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, 1);
    delete op_;

    if (a)
    {
      h_index index_av = h_index (index_a);
      index_tmp.setVT();
      index_av.setVT();

      op_ = new h_ops_tree (gemm, &index_tmp, &index_av, index_b);
    }
    else
    {
      h_index index_bu = h_index (index_b);
      index_tmp.setU();
      index_bu.setU();

      op_ = new h_ops_tree (gemm, &index_tmp, index_a, &index_bu);
    }

    op -> setChild (op_, 0);
    delete op_;
  }

  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Hierarchical * A, const h_index * index_a, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  const int n_k = A -> getNx_blocks(), n_mk = n_k * A -> getNy_blocks();
  int * y, * k, x = index_b -> getNx(self -> getNx());
  A -> getOffsets_y(&y);
  A -> getOffsets_x(&k);

  if (self -> isU() && index_b -> isU())
  {
      op -> resizeChildren (n_mk);

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mk; i++)
    {
      const int row = i / n_k, col = i - row * n_k;
      const h_index index_ai = h_index (A, index_a, row, col), index_m = h_index (self, y[row], 0, index_ai.getNy(), x), index_bj = h_index (index_b, k[col], 0, index_ai.getNx(), x);
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A -> getElement_blocks(row, col), &index_ai, B, &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }
  else
  {
    op -> resizeChildren (n_mk + 1);

    int rank_b = index_b -> getRank(), tmp_size = rank_b * index_a -> getNy(self -> getNy()), block_id;
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self), index_bu = h_index (index_b); 
    index_tmp.setTemp_Low_Rank (block_id, rank_b);
    index_tmp.setVT_data (index_b);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, n_mk);
    delete op_;

    index_tmp.setU();
    index_bu.setU();

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mk; i++)
    {
      const int row = i / n_k, col = i - row * n_k;
      const h_index index_ai = h_index (A, index_a, row, col), index_m = h_index (&index_tmp, y[row], 0, index_ai.getNy(), x), index_bj = h_index (&index_bu, k[col], 0, index_ai.getNx(), x);
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A -> getElement_blocks(row, col), &index_ai, B, &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }

  delete[] y;
  delete[] k;
  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Element * A, const h_index * index_a, const LowRank * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_a = A -> getElementHierarchical();
  const LowRank *lr_a = A -> getElementLowRank();
  const Dense *d_a = A -> getElementDense();

  if (d_a != nullptr)
  { return generateOps_GEMM (self, d_a, index_a, B, index_b, tmp_mngr); }
  if (lr_a != nullptr)
  { return generateOps_GEMM (self, lr_a, index_a, B, index_b, tmp_mngr); }
  if (h_a != nullptr)
  { return generateOps_GEMM (self, h_a, index_a, B, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Dense * A, const h_index * index_a, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  const int n_n = B -> getNx_blocks(), n_nk = n_n * B -> getNy_blocks();
  int * x, * k, y = self -> getNy(index_a -> getNy());
  B -> getOffsets_y(&k);
  B -> getOffsets_x(&x);

  if (self -> isLowRank_Full())
  {
    printf("WARNING: Potential Accuracy Loss from an attempt to accumulate Dense into Low-Rank.\n");

    op -> resizeChildren (n_nk + 1);

    int block_id, tmp_size = self -> getSize();
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self); index_tmp.setTemp_Dense (block_id);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, n_nk);
    delete op_;

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_nk; i++)
    {
      const int row = i / n_n, col = i - row * n_n;
      const h_index index_bj = h_index (B, index_b, row, col), index_m = h_index (&index_tmp, 0, x[col], y, index_bj.getNx()), index_ai = h_index (index_a, 0, k[row], y, index_bj.getNy());
      h_ops_tree * op_i = Dense::generateOps_GEMM(&index_m, A, &index_ai, B -> getElement_blocks(row, col), &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }
  else
  {
    op -> resizeChildren (n_nk);

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_nk; i++)
    {
      const int row = i / n_n, col = i - row * n_n;
      const h_index index_bj = h_index (B, index_b, row, col), index_m = h_index (self, 0, x[col], y, index_bj.getNx()), index_ai = h_index (index_a, 0, k[row], y, index_bj.getNy());
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A, &index_ai, B -> getElement_blocks(row, col), &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }

  delete[] x;
  delete[] k;
  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const LowRank * A, const h_index * index_a, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  const int n_n = B -> getNx_blocks(), n_nk = n_n * B -> getNy_blocks();
    
  int * x, * k, y = self -> getNy(index_a -> getNy());
  B -> getOffsets_x(&x);
  B -> getOffsets_y(&k);

  if (self -> isVT() && index_a -> isVT())
  {
    op -> resizeChildren(n_nk);

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_nk; i++)
    {
      const int row = i / n_n, col = i - row * n_n;
      const h_index index_bj = h_index (B, index_b, row, col), index_m = h_index (self, 0, x[col], y, index_bj.getNx()), index_ai = h_index (index_a, 0, k[row], y, index_bj.getNy());
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A, &index_ai, B -> getElement_blocks(row, col), &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }
  else
  {
    op -> resizeChildren(n_nk + 1);

    int rank_a = index_a -> getRank(), tmp_size = rank_a * index_b -> getNx(self -> getNx()), block_id;
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self), index_av = h_index (index_a); 
    index_tmp.setTemp_Low_Rank(block_id, rank_a);
    index_tmp.setU_data(index_a);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, n_nk);
    delete op_;

    index_tmp.setVT();
    index_av.setVT();

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_nk; i++)
    {
      const int row = i / n_n, col = i - row * n_n;
      const h_index index_bj = h_index (B, index_b, row, col), index_m = h_index (&index_tmp, 0, x[col], y, index_bj.getNx()), index_ai = h_index (&index_av, 0, k[row], y, index_bj.getNy());
      h_ops_tree * op_i = generateOps_GEMM(&index_m, A, &index_ai, B -> getElement_blocks(row, col), &index_bj, tmp_mngr);
      op -> setChild(op_i, i);
      delete op_i;
    }
  }

  delete[] x;
  delete[] k;
  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Hierarchical * A, const h_index * index_a, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const int n_k = A -> getNx_blocks(); if (n_k != B -> getNy_blocks())
  { printf("Matrices are partitioned differently in LR.H-H GEMM.\n"); return nullptr; }

  h_ops_tree * op = new h_ops_tree (gemm, self, index_a, index_b);

  const int n_n = B -> getNx_blocks(), n_mn = n_n * A -> getNy_blocks(), n_mnk = n_mn * n_k;
  int * x, * y;
  A -> getOffsets_y(&y);
  B -> getOffsets_x(&x);

  if (self -> isLowRank_Full())
  {
    printf("WARNING: Potential Accuracy Loss from an attempt to accumulate Dense into Low-Rank.\n");

    op -> resizeChildren(n_mnk + 1);

    int block_id, tmp_size = self -> getSize();
#pragma omp critical
    { block_id = tmp_mngr -> requestTemp(tmp_size); }

    h_index index_tmp = h_index (self); index_tmp.setTemp_Dense (block_id);

    h_ops_tree * op_ = new h_ops_tree (accum, self, &index_tmp);
    op -> setChild (op_, n_mnk);
    delete op_;

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mnk; i++)
    {
      const int k = i / n_mn, crd = i - k * n_mn, row = crd / n_n, col = crd - row * n_n;

      const h_index index_ai = h_index (A, index_a, row, k), index_bj = h_index (B, index_b, k, col);
      const h_index index_m = h_index (&index_tmp, y[row], x[col], index_ai.getNy(), index_bj.getNx());
      h_ops_tree * op_k = Dense::generateOps_GEMM (&index_m, A -> getElement_blocks(row, k), &index_ai, B -> getElement_blocks(k, col), &index_bj, tmp_mngr);
      op -> setChild(op_k, i);
      delete op_k;
    }
  }
  else
  {
    op -> resizeChildren(n_mnk);

#pragma omp parallel for if (omp_in_parallel() == 0)
    for (int i = 0; i < n_mnk; i++)
    {
      const int k = i / n_mn, crd = i - k * n_mn, row = crd / n_n, col = crd - row * n_n;

      const h_index index_ai = h_index (A, index_a, row, k), index_bj = h_index (B, index_b, k, col);
      const h_index index_m = h_index (self, y[row], x[col], index_ai.getNy(), index_bj.getNx());
      h_ops_tree * op_k = generateOps_GEMM (&index_m, A -> getElement_blocks(row, k), &index_ai, B -> getElement_blocks(k, col), &index_bj, tmp_mngr);
      op -> setChild(op_k, i);
      delete op_k;
    }
  }

  delete[] x;
  delete[] y;
  return op;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Element * A, const h_index * index_a, const Hierarchical * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_a = A -> getElementHierarchical();
  const LowRank *lr_a = A -> getElementLowRank();
  const Dense *d_a = A -> getElementDense();

  if (d_a != nullptr)
  { return generateOps_GEMM (self, d_a, index_a, B, index_b, tmp_mngr); }
  if (lr_a != nullptr)
  { return generateOps_GEMM (self, lr_a, index_a, B, index_b, tmp_mngr); }
  if (h_a != nullptr)
  { return generateOps_GEMM (self, h_a, index_a, B, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Dense * A, const h_index * index_a, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_b = B -> getElementHierarchical();
  const LowRank *lr_b = B -> getElementLowRank();
  const Dense *d_b = B -> getElementDense();

  if (d_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, d_b, index_b, tmp_mngr); }
  if (lr_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, lr_b, index_b, tmp_mngr); }
  if (h_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, h_b, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const LowRank * A, const h_index * index_a, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_b = B -> getElementHierarchical();
  const LowRank *lr_b = B -> getElementLowRank();
  const Dense *d_b = B -> getElementDense();

  if (d_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, d_b, index_b, tmp_mngr); }
  if (lr_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, lr_b, index_b, tmp_mngr); }
  if (h_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, h_b, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Hierarchical * A, const h_index * index_a, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_b = B -> getElementHierarchical();
  const LowRank *lr_b = B -> getElementLowRank();
  const Dense *d_b = B -> getElementDense();

  if (d_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, d_b, index_b, tmp_mngr); }
  if (lr_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, lr_b, index_b, tmp_mngr); }
  if (h_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, h_b, index_b, tmp_mngr); }

  return nullptr;
}

h_ops_tree * LowRank::generateOps_GEMM (const h_index * self, const Element * A, const h_index * index_a, const Element * B, const h_index * index_b, dev_temp * tmp_mngr)
{
  const Hierarchical *h_b = B -> getElementHierarchical();
  const LowRank *lr_b = B -> getElementLowRank();
  const Dense *d_b = B -> getElementDense();

  if (d_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, d_b, index_b, tmp_mngr); }
  if (lr_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, lr_b, index_b, tmp_mngr); }
  if (h_b != nullptr)
  { return generateOps_GEMM (self, A, index_a, h_b, index_b, tmp_mngr); }

  return nullptr;
}*/

