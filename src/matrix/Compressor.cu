#include "hip/hip_runtime.h"

#include <matrix/Compressor.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <omp.h>

const int n_stream = 4;

compressor::compressor(Hierarchical& h, const int rank, const double condition) {
  load(h, rank, condition);
  compress(rank);
}

compressor::~compressor() {
  for (auto iter = d_lis.begin(); iter != d_lis.end(); iter++) {
    delete *iter;
  }
}

void compressor::load(Hierarchical& h, const int rank, const double condition) {
  for (int x = 0; x < h.getPartX(); x++) {
    for (int y = 0; y < h.getPartY(); y++) {
      Element* e = h.getChild(y, x);
      if (e != nullptr && e->getElementHierarchical() != nullptr) {
        load(*(e->getElementHierarchical()), rank, condition);
      }
      else if (e != nullptr && e->getElementDense() != nullptr) {
        bool admis = e->admissible(condition);
        if (admis) {
          d_lis.push_back(e->getElementDense());
          LowRank* lr = new LowRank(e->getRowDimension(), e->getColumnDimension(), rank);
          lr_lis.push_back(lr);
          h.setElement(lr, y, x);
        }
      }
    }
  }
}

void compressor::compress(const int rank) {
  hipStream_t streams[n_stream];
  hipsolverHandle_t shandle[n_stream];
  hipblasHandle_t chandle[n_stream];
  real_t* workspace[n_stream];

  for (int i = 0; i < n_stream; i++) {
    hipStreamCreate(&streams[i]);
    hipsolverDnCreate(&shandle[i]);
    hipblasCreate(&chandle[i]);
    hipsolverSetStream(shandle[i], streams[i]);
    hipblasSetStream(chandle[i], streams[i]);
    hipMalloc(reinterpret_cast<void**>(&workspace[i]), 16384);
    hipblasSetWorkspace(chandle[i], workspace[i], 16384);
  }

  int size = (int)d_lis.size() / n_stream;
#pragma omp parallel num_threads(n_stream) 
  {
    int tid = omp_get_thread_num();
    int start_i = tid * size;
    using std::min;
    int end_i = min((int)d_lis.size(), start_i + size);

    for (int i = start_i; i < end_i; i++) {
      LowRank* lr = lr_lis[i]->getElementLowRank();
      real_t* Q = lr->getU()->getElements();
    }
#pragma omp critical
    std::cout << tid << ": " << start_i << " " << end_i << std::endl;
  }

  for (int i = 0; i < n_stream; i++) {
    hipsolverDnDestroy(shandle[i]);
    hipblasDestroy(chandle[i]);
    hipFree(workspace[i]);
  }

}
