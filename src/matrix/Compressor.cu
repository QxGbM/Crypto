
#include <matrix/Compressor.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <omp.h>

const int n_stream = 4;

compressor::compressor(Hierarchical& h, const int rank, const double condition) {
  load(h, rank, condition);
  compress();
}

compressor::~compressor() {
  for (auto iter = d_lis.begin(); iter != d_lis.end(); iter++) {
    delete* iter;
  }
}

void compressor::load(Hierarchical& h, const int rank, const double condition) {
  for (int x = 0; x < h.getPartX(); x++) {
    for (int y = 0; y < h.getPartY(); y++) {
      Element* e = h.getChild(y, x);
      if (e != nullptr && e->getElementHierarchical() != nullptr) {
        load(*(e->getElementHierarchical()), rank, condition);
      }
      else if (e != nullptr && e->getElementDense() != nullptr) {
        bool admis = e->admissible(condition);
        if (admis) {
          d_lis.push_back(e->getElementDense());
          LowRank* lr = new LowRank(e->getRowDimension(), e->getColumnDimension(), rank);
          lr_lis.push_back(lr);
          h.setElement(lr, y, x);
        }
      }
    }
  }
}

void compressor::compress() {
  hipStream_t streams[n_stream];
  hipsolverHandle_t shandle[n_stream];
  hipblasHandle_t chandle[n_stream];
  real_t* workspace[n_stream];

  for (int i = 0; i < n_stream; i++) {
    hipStreamCreate(&streams[i]);
    hipsolverDnCreate(&shandle[i]);
    hipblasCreate(&chandle[i]);
    hipsolverSetStream(shandle[i], streams[i]);
    hipblasSetStream(chandle[i], streams[i]);
    hipMalloc(reinterpret_cast<void**>(&workspace[i]), 16384);
    hipblasSetWorkspace(chandle[i], workspace[i], 16384);
  }

#pragma omp parallel for num_threads(n_stream)
  for (int i = 0; i < d_lis.size(); i++) {
    int tid = omp_get_thread_num();

  }

  for (int i = 0; i < n_stream; i++) {
    hipsolverDnDestroy(shandle[i]);
    hipblasDestroy(chandle[i]);
    hipFree(workspace[i]);
  }

}
