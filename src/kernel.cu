#include "hip/hip_runtime.h"

#include <definitions.cuh>
#include <kernel.cuh>

DEVICE int thread_rank()
{ return (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x; }

DEVICE int block_dim()
{ return blockDim.z * blockDim.y * blockDim.x; }

DEVICE int block_rank()
{ return (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x; }

DEVICE int grid_dim()
{ return gridDim.z * gridDim.y * gridDim.x; }

DEVICE int warp_rank()
{
  unsigned int warpid;
  asm volatile("mov.u32 %0, %warpid;" : "=r"(warpid));
  return (int) warpid;
}

DEVICE int lane_rank()
{ 
  unsigned int laneid;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(laneid));
  return (int) laneid;
}

DEVICE int num_warps()
{ return (block_dim() + warpSize - 1) / warpSize; }

DEVICE void wait (clock_t lapse)
{
  clock_t start = clock64();
  while (lapse > abs(clock64() - start));
  return;
}

/* A convinient call to copy from shared memory to global or vice versa. Reading "from" in row major. */
DEVICE void matrixCopy (const real_t * __restrict__ from, real_t * __restrict__ to, const int nx_to, const int ny_to, const int ld_from, const int ld_to)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps();
  const int iter = nx_to / vec_size, last_start = iter * vec_size, last = nx_to - last_start;

  for (int row = w_id; row < ny_to; row += n_wp)
  {
    real_t * to_row = &to[row * ld_to];
    const real_t * from_row = &from[row * ld_from];

    for (int col = l_id; col < iter; col += warpSize)
    { reinterpret_cast <vec_t *> (to_row)[col] = reinterpret_cast <const vec_t *> (from_row)[col]; }
  }

  if (last > 0)
  for (int row = w_id; row < ny_to; row += n_wp)
  {
    real_t * to_row = &to[row * ld_to + last_start];
    const real_t * from_row = &from[row * ld_from + last_start];

    for (int col = l_id; col < last; col += warpSize)
    { to_row[col] = from_row[col]; }
  }

}

/* A convinient call to copy from shared memory to global or vice versa. Reading "from" in row major. */
DEVICE int matrixCopy_keepT (const real_t * __restrict__ from, real_t * __restrict__ to, const int nx_from, const int ny_from, const int ld_from, const bool transpose)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps();

  const int nx_real = transpose ? ny_from : nx_from, ny_real = transpose ? nx_from : ny_from;
  const int ld_to = ((nx_real + vec_size - 1) / vec_size) * vec_size;

  const int iter = nx_real / vec_size, last_start = iter * vec_size, last = nx_real - iter * vec_size;

  for (int row = w_id; row < ny_real; row += n_wp)
  {
    real_t * to_row = &to[row * ld_to];
    const real_t * from_row = &from[row * ld_from];

    for (int col = l_id; col < iter; col += warpSize)
    { reinterpret_cast <vec_t *> (to_row)[col] = reinterpret_cast <const vec_t *> (from_row)[col]; }
  }

  if (last > 0)
  for (int row = w_id; row < ny_real; row += n_wp)
  {
    real_t * to_row = &to[row * ld_to + last_start];
    const real_t * from_row = &from[row * ld_from + last_start];

    for (int col = l_id; col < last; col += warpSize)
    { to_row[col] = from_row[col]; }
  }

  return ld_to;

}


/* LU decomposition of matrix of ny by nx. */
DEVICE void DenseGetrf (real_t * M, const int nx, const int ny, const int ld)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps(), min_n = nx > ny ? ny : nx;

  real_t left, vec0[vec_size], vec1[vec_size];
  
  for (int i = 0; i < min_n; i ++)
  {
    const int x_start = i + 1, x_n = nx - x_start;

    const int iter = x_n / vec_size, last = x_n - iter * vec_size, align = x_start + last;

    real_t * M_top = &M[i * ld], * M_top_align = &M_top[align];

    for (int row = w_id + i + 1; row < ny; row += n_wp)
    {
      real_t * M_row = &M[row * ld], * M_row_align = &M_row[align];

      if (l_id == 0)
      { M_row[i] = left = M_row[i] / M_top[i]; }

      left = __shfl_sync (0xffffffff, - left, 0, warpSize);

      if (last > 0)
      for (int col = x_start + l_id; col < align; col += warpSize)
      { M_row[col] = fma (M_top[col], left, M_row[col]); }

      for (int col = l_id; col < iter; col += warpSize)
      {
        reinterpret_cast <vec_t *> (vec0)[0] = reinterpret_cast <vec_t *> (M_top_align)[col];
        reinterpret_cast <vec_t *> (vec1)[0] = reinterpret_cast <vec_t *> (M_row_align)[col];

        #pragma unroll
        for (int i1 = 0; i1 < vec_size; i1++)
        { vec1[i1] = fma (vec0[i1], left, vec1[i1]); }

        reinterpret_cast <vec_t *> (M_row_align)[col] = reinterpret_cast <vec_t *> (vec1)[0];
      }

    }
    __syncthreads();
  }

}

/* L is ny_l x nx_l lower triangular and unit diagonal, B is ny_l by nx_b, solves L x X = B, overwrites X in B. */
DEVICE void DenseTrsmL (real_t * __restrict__ B, const real_t * __restrict__ L, const int nx_b, const int ny_b, const int nx_l, const int ld_b, const int ld_l)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps(), min_n = nx_l > ny_b ? ny_b : nx_l;

  const int iter = nx_b / vec_size, serial_start = iter * vec_size, last = nx_b - serial_start;
  real_t vec0[vec_size], vec1[vec_size];

  for (int i = 0; i < min_n; i ++)
  {
    for (int row = w_id + i + 1; row < ny_b; row += n_wp)
    {
      real_t left = - L[row * ld_l + i], * B_top = &B[i * ld_b], * B_row = &B[row * ld_b];

      for (int col = l_id; col < iter; col += warpSize)
      {
        reinterpret_cast <vec_t *> (vec0)[0] = reinterpret_cast <vec_t *> (B_top)[col];
        reinterpret_cast <vec_t *> (vec1)[0] = reinterpret_cast <vec_t *> (B_row)[col];

        #pragma unroll
        for (int i1 = 0; i1 < vec_size; i1++)
        { vec1[i1] = fma (vec0[i1], left, vec1[i1]); }

        reinterpret_cast <vec_t *> (B_row)[col] = reinterpret_cast <vec_t *> (vec1)[0];
      }

      if (last > 0)
      for (int col = serial_start + l_id; col < nx_b; col += warpSize)
      { B_row[col] = fma (B_top[col], left, B_row[col]); }
    }
    __syncthreads();
  }

}

/* U is ny_u x nx_u upper triangular and not unit diagonal, B is ny_b by nx_u, solves X x U = B, overwrites X in B. */
DEVICE void DenseTrsmR (real_t * __restrict__ B, const real_t * __restrict__ U, const int nx_b, const int ny_b, const int ny_u, const int ld_b, const int ld_u)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps(), min_n = nx_b > ny_u ? ny_u : nx_b;

  real_t left, vec0[vec_size], vec1[vec_size];

  for (int i = 0; i < min_n; i ++)
  {
    const int i_start = i + 1, x = nx_b - i_start;
    const int iter = x / vec_size, last = x - iter * vec_size, last_start = i_start + last;

    const real_t * U_top = &U[i * ld_u], * U_top_vec = &U_top[last_start];

    for (int row = w_id; row < ny_b; row += n_wp)
    {
      real_t * B_row = &B[row * ld_b], * B_row_vec = &B_row[last_start];

      if (l_id == 0)
      { B_row[i] = left = B_row[i] / U_top[i]; }

      left = __shfl_sync (0xffffffff, - left, 0, warpSize);

      if (last > 0)
      for (int col = l_id + i_start; col < last_start; col += warpSize)
      { B_row[col] = fma (left, U_top[col], B_row[col]); }

      for (int col = l_id; col < iter; col += warpSize)
      {
        reinterpret_cast <vec_t *> (vec0)[0] = reinterpret_cast <const vec_t *> (U_top_vec)[col];
        reinterpret_cast <vec_t *> (vec1)[0] = reinterpret_cast <vec_t *> (B_row_vec)[col];

        #pragma unroll
        for (int i1 = 0; i1 < vec_size; i1++)
        { vec1[i1] = fma (vec0[i1], left, vec1[i1]); }

        reinterpret_cast <vec_t *> (B_row_vec)[col] = reinterpret_cast <vec_t *> (vec1)[0];
      }
    }
    __syncthreads();
  }

}

/* U is ny_u x nx_u upper triangular and not unit diagonal, B is ny_b by nx_u, solves X x U = B, overwrites X in B. */
DEVICE void DenseTrsmR_transposeB (real_t * __restrict__ B, const real_t * __restrict__ U, const int nx_b, const int ny_b, const int ny_u, const int ld_b, const int ld_u)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps(), min_n = nx_b > ny_u ? ny_u : nx_b;

  const int iter = ny_b / vec_size, serial_start = iter * vec_size, last = ny_b - serial_start;
  real_t vec0[vec_size], vec1[vec_size];

  for (int i = 0; i < min_n; i ++)
  {
    real_t * B_top = &B[i * ld_b];
    const real_t * U_top = &U[i * ld_u];

    if (w_id == 0)
    {
      real_t diag = 1. / U_top[i];

      for (int col = l_id; col < iter; col += warpSize)
      { 
        reinterpret_cast <vec_t *> (vec1)[0] = reinterpret_cast <vec_t *> (B_top)[col];

        #pragma unroll
        for (int i1 = 0; i1 < vec_size; i1++)
        { vec1[i1] *= diag; }

        reinterpret_cast <vec_t *> (B_top)[col] = reinterpret_cast <vec_t *> (vec1)[0];
      }

      if (last > 0)
      for (int col = l_id + serial_start; col < ny_b; col += warpSize)
      { B_top[col] *= diag; }
    }
    __syncthreads();

    for (int row = w_id + i + 1; row < nx_b; row += n_wp)
    {
      real_t left = - U_top[row], * B_row = &B[row * ld_b];

      for (int col = l_id; col < iter; col += warpSize)
      { 
        reinterpret_cast <vec_t *> (vec0)[0] = reinterpret_cast <vec_t *> (B_top)[col];
        reinterpret_cast <vec_t *> (vec1)[0] = reinterpret_cast <vec_t *> (B_row)[col];

        #pragma unroll
        for (int i1 = 0; i1 < vec_size; i1++)
        { vec1[i1] = fma (left, vec0[i1], vec1[i1]); }

        reinterpret_cast <vec_t *> (B_row)[col] = reinterpret_cast <vec_t *> (vec1)[0];
      }

      if (last > 0)
      for (int col = l_id + serial_start; col < ny_b; col += warpSize)
      { B_row[col] = fma (left, B_top[col], B_row[col]); }
    }
    __syncthreads();
  }

}

/* General Matrix multiplication. M (m by n) = A (m by k) * B (k by n) + old_M. */
DEVICE void DenseGemm (real_t * __restrict__ M, const real_t * __restrict__ A, const real_t * __restrict__ B, const int m, const int n, const int k, 
  const int ld_m, const int ld_a, const int ld_b, const bool a_T, const bool b_T)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps();

  int A_step, B_step, A_iter, B_iter;

  if (a_T) { A_step = 1; A_iter = ld_a; } else { A_step = ld_a; A_iter = 1; }
  if (b_T) { B_step = ld_b; B_iter = 1; } else { B_step = 1; B_iter = ld_b; }

  const int iter_m = m / vec_size, iter_n = n / vec_size;
  const int last_m_start = iter_m * vec_size, last_n_start = iter_n * vec_size;
  const int last_m = m - last_m_start, last_n = n - last_n_start;

  const bool b_last_m = last_m - w_id > 0, b_last_n = last_n > 0;

  real_t thread_a[vec_size], thread_b[vec_size], thread_m[vec_size][vec_size];

  const real_t * A_k = A, * B_k = B;

  for (int i = 0; i < k; i++)
  {
    for (int i1 = w_id; i1 < iter_m; i1 += n_wp)
    {
      const int row_start = i1 * vec_size;

      #pragma unroll
      for (int i3 = 0; i3 < vec_size; i3++)
      {
        const int row = row_start + i3;
        thread_a[i3] = A_k[row * A_step]; 
      }

      for (int i2 = l_id; i2 < iter_n; i2 += warpSize)
      {
        const int col_start = i2 * vec_size;

        #pragma unroll
        for (int i3 = 0; i3 < vec_size; i3++)
        {
          const int row = row_start + i3;
          reinterpret_cast <vec_t *> (thread_m[i3])[0] = reinterpret_cast <vec_t *> (&M[row * ld_m])[i2];
        }

        #pragma unroll
        for (int i4 = 0; i4 < vec_size; i4++)
        {
          const int col = col_start + i4;
          thread_b[i4] = B_k[col * B_step];
        }

        #pragma unroll
        for (int i3 = 0; i3 < vec_size; i3++)
        {
          #pragma unroll
          for (int i4 = 0; i4 < vec_size; i4++)
          { thread_m[i3][i4] = fma (thread_a[i3], thread_b[i4], thread_m[i3][i4]); }
        }

        #pragma unroll
        for (int i3 = 0; i3 < vec_size; i3++)
        {
          const int row = row_start + i3;
          reinterpret_cast <vec_t *> (&M[row * ld_m])[i2] = reinterpret_cast <vec_t *> (thread_m[i3])[0];
        }

      }

      if (b_last_n)
      {
        #pragma unroll
        for (int i3 = 0; i3 < vec_size; i3++)
        {
          const int row = row_start + i3;
          real_t * M_row = &M[row * ld_m + last_n_start];

          for (int col = l_id; col < last_n; col += warpSize)
          { M_row[col] = fma (thread_a[i3], B_k[col * B_step], M_row[col]); }
        }
      }

    }

    if (b_last_m)
    {
      const int row = w_id + last_m_start; const real_t left = A_k[row * A_step];
      real_t * M_row = &M[row * ld_m];

      for (int i2 = l_id; i2 < iter_n; i2 += warpSize)
      {
        const int col_start = i2 * vec_size;
        reinterpret_cast <vec_t *> (thread_a)[0] = reinterpret_cast <vec_t *> (M_row)[i2];

        #pragma unroll
        for (int i4 = 0; i4 < vec_size; i4++)
        {
          const int col = col_start + i4;
          thread_b[i4] = B_k[col * B_step]; 
        }

        #pragma unroll
        for (int i4 = 0; i4 < vec_size; i4++)
        { thread_a[i4] = fma(left, thread_b[i4], thread_a[i4]); }

        reinterpret_cast <vec_t *> (M_row)[i2] = reinterpret_cast <vec_t *> (thread_a)[0];
      }

      if (b_last_n)
      {
        real_t * M_row_n = &M_row[last_n_start];

        #pragma unroll
        for (int i3 = 0; i3 < vec_size; i3++)
        {
          for (int col = l_id; col < last_n; col += warpSize)
          { M_row_n[col] = fma (left, B_k[col * B_step], M_row_n[col]); }
        }
      }
    }

    A_k = &A_k[A_iter];
    B_k = &B_k[B_iter];
  }

}

/* General Matrix multiplication. M (m by n) = alpha * A (m by k) * B (k by n) + beta * old_M. */
DEVICE void blockDenseGemm (const real_t alpha, const real_t beta, real_t * __restrict__ M, const real_t * __restrict__ A, const real_t * __restrict__ B,
  const int m, const int n, const int k, const int ld_m, const int ld_a, const int ld_b, const bool a_T, const bool b_T, real_t * __restrict__ shm)
{
  const int w_id = warp_rank(), l_id = lane_rank(), n_wp = num_warps();

  const int iter_n = n / vec_size;
  const int last_n_start = iter_n * vec_size;
  const int last_n = n - last_n_start;

  const bool b_last_n = last_n > 0;

  real_t mult = beta / alpha;

  if (beta == 0.)
  {
    real_t thread_a[vec_size];
    #pragma unroll
    for (int i = 0; i < vec_size; i++)
    { thread_a[i] = 0.; }
    
    vec_t zero_vec = reinterpret_cast <vec_t *> (thread_a)[0];

    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m];

      for (int col = l_id; col < iter_n; col += warpSize)
      { reinterpret_cast <vec_t *> (M_row)[col] = zero_vec; }
    }

    if (b_last_n)
    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m + last_n_start];

      for (int col = l_id; col < last_n; col += warpSize)
      { M_row[col] = 0.; }
    }
    __syncthreads();
  }
  else if (mult != 1.)
  {
    real_t thread_a[vec_size];

    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m];

      for (int col = l_id; col < iter_n; col += warpSize)
      {
        reinterpret_cast <vec_t *> (thread_a)[0] = reinterpret_cast <vec_t *> (M_row)[col];

        #pragma unroll
        for (int i = 0; i < vec_size; i++)
        { thread_a[i] *= mult; }

        reinterpret_cast <vec_t *> (M_row)[col] = reinterpret_cast <vec_t *> (thread_a)[0];
      }
    }

    if (b_last_n)
    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m + last_n_start];

      for (int col = l_id; col < last_n; col += warpSize)
      { M_row[col] *= mult; }
    }
    __syncthreads();
  }

  real_t * shm_a = &shm[_BLOCK_M * _BLOCK_M], * shm_b = &shm[_BLOCK_M * (_BLOCK_M + _BLOCK_K)];

  const int iters_m = m / _BLOCK_M, iters_n = n / _BLOCK_M, iters_k = k / _BLOCK_K;
  const int last_m_dim = m - iters_m * _BLOCK_M, last_n_dim = n - iters_n * _BLOCK_M;
  const int last_k_dim = k - iters_k * _BLOCK_K;

  int A_step_r, B_step_r, A_step_c, B_step_c;

  if (a_T) { A_step_r = 1; A_step_c = ld_a; } else { A_step_r = ld_a; A_step_c = 1; }
  if (b_T) { B_step_r = 1; B_step_c = ld_b; } else { B_step_r = ld_b; B_step_c = 1; }

  for (int i0 = 0; i0 < iters_m; i0++)
  {
    const int m_off = i0 * _BLOCK_M;
    real_t * M_row = &M[m_off * ld_m]; const real_t * A_row = &A[m_off * A_step_r];

    for (int i1 = 0; i1 < iters_n; i1++)
    {
      const int n_off = i1 * _BLOCK_M; const real_t * B_col = &B[n_off * B_step_c];
      const int ld_0 = matrixCopy_keepT (&M_row[n_off], shm, _BLOCK_M, _BLOCK_M, ld_m, false);

      for (int i2 = 0; i2 < iters_k; i2++)
      {
        const int k_off = i2 * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, _BLOCK_K, _BLOCK_M, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, _BLOCK_M, _BLOCK_K, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, _BLOCK_M, _BLOCK_M, _BLOCK_K, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      if (last_k_dim > 0)
      {
        const int k_off = iters_k * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, last_k_dim, _BLOCK_M, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, _BLOCK_M, last_k_dim, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, _BLOCK_M, _BLOCK_M, last_k_dim, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      matrixCopy (shm, &M_row[n_off], _BLOCK_M, _BLOCK_M, ld_0, ld_m);
      __syncthreads();
    }

    if (last_n_dim > 0)
    {
      const int n_off = iters_n * _BLOCK_M; const real_t * B_col = &B[n_off * B_step_c];
      const int ld_0 = matrixCopy_keepT (&M_row[n_off], shm, last_n_dim, _BLOCK_M, ld_m, false);

      for (int i2 = 0; i2 < iters_k; i2++)
      {
        const int k_off = i2 * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, _BLOCK_K, _BLOCK_M, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, last_n_dim, _BLOCK_K, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, _BLOCK_M, last_n_dim, _BLOCK_K, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      if (last_k_dim > 0)
      {
        const int k_off = iters_k * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, last_k_dim, _BLOCK_M, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, last_n_dim, last_k_dim, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, _BLOCK_M, last_n_dim, last_k_dim, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      matrixCopy (shm, &M_row[n_off], last_n_dim, _BLOCK_M, ld_0, ld_m);
      __syncthreads();
    }
  }

  if (last_m_dim > 0)
  {
    const int m_off = iters_m * _BLOCK_M;
    real_t * M_row = &M[m_off * ld_m]; const real_t * A_row = &A[m_off * A_step_r];

    for (int i1 = 0; i1 < iters_n; i1++)
    {
      const int n_off = i1 * _BLOCK_M; const real_t * B_col = &B[n_off * B_step_c];
      const int ld_0 = matrixCopy_keepT (&M_row[n_off], shm, _BLOCK_M, last_m_dim, ld_m, false);

      for (int i2 = 0; i2 < iters_k; i2++)
      {
        const int k_off = i2 * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, _BLOCK_K, last_m_dim, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, _BLOCK_M, _BLOCK_K, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, last_m_dim, _BLOCK_M, _BLOCK_K, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      if (last_k_dim > 0)
      {
        const int k_off = iters_k * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, last_k_dim, last_m_dim, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, _BLOCK_M, last_k_dim, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, last_m_dim, _BLOCK_M, last_k_dim, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      matrixCopy (shm, &M_row[n_off], _BLOCK_M, last_m_dim, ld_0, ld_m);
      __syncthreads();
    }

    if (last_n_dim > 0)
    {
      const int n_off = iters_n * _BLOCK_M; const real_t * B_col = &B[n_off * B_step_c];
      const int ld_0 = matrixCopy_keepT (&M_row[n_off], shm, last_n_dim, _BLOCK_M, ld_m, false);

      for (int i2 = 0; i2 < iters_k; i2++)
      {
        const int k_off = i2 * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, _BLOCK_K, last_m_dim, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, last_n_dim, _BLOCK_K, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, last_m_dim, last_n_dim, _BLOCK_K, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      if (last_k_dim > 0)
      {
        const int k_off = iters_k * _BLOCK_K; const real_t * A_k = &A_row[k_off * A_step_c], * B_k = &B_col[k_off * B_step_r];
        const int ld_1 = matrixCopy_keepT (A_k, shm_a, last_k_dim, last_m_dim, ld_a, a_T);
        const int ld_2 = matrixCopy_keepT (B_k, shm_b, last_n_dim, last_k_dim, ld_b, b_T);
        __syncthreads();

        DenseGemm (shm, shm_a, shm_b, last_m_dim, last_n_dim, last_k_dim, ld_0, ld_1, ld_2, a_T, b_T);
        __syncthreads();
      }

      matrixCopy (shm, &M_row[n_off], last_n_dim, last_m_dim, ld_0, ld_m);
      __syncthreads();
    }
  }

  if (alpha != 1.)
  {
    real_t thread_a[vec_size];

    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m];

      for (int col = l_id; col < iter_n; col += warpSize)
      {
        reinterpret_cast <vec_t *> (thread_a)[0] = reinterpret_cast <vec_t *> (M_row)[col];

        #pragma unroll
        for (int i = 0; i < vec_size; i++)
        { thread_a[i] *= alpha; }

        reinterpret_cast <vec_t *> (M_row)[col] = reinterpret_cast <vec_t *> (thread_a)[0];
      }
    }

    if (b_last_n)
    for (int row = w_id; row < m; row += n_wp)
    {
      real_t * M_row = &M[row * ld_m + last_n_start];

      for (int col = l_id; col < last_n; col += warpSize)
      { M_row[col] *= alpha; }
    }
  }
  __syncthreads();

}

/* L is ny_l x nx_l lower triangular and unit diagonal, B is ny_l by nx_b, solves L x X = B, overwrites X in B. */
DEVICE void blockDenseTrsmL (real_t * __restrict__ B, const real_t * __restrict__ L, const int nx_b, const int ny_b, const int nx_l, const int ld_b, const int ld_l, real_t * __restrict__ shm)
{
  const int l_step = _BLOCK_M * ld_l + _BLOCK_M, b_step = _BLOCK_M * ld_b; int remain_nx = nx_l, remain_ny = ny_b;
  const real_t * L_diag = L, * L_left = &L[l_step - _BLOCK_M];
  real_t * B_top = B, * B_next = &B[b_step];

  while (remain_nx > _BLOCK_M && remain_ny > _BLOCK_M)
  {
    remain_nx -= _BLOCK_M;
    remain_ny -= _BLOCK_M;

    const int ld_0 = matrixCopy_keepT (L_diag, shm, _BLOCK_M, _BLOCK_M, ld_l, false);
    __syncthreads();

    DenseTrsmL (B_top, shm, nx_b, _BLOCK_M, _BLOCK_M, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, L_left, B_top, remain_ny, nx_b, _BLOCK_M, ld_b, ld_l, ld_b, false, false, shm);

    L_diag = &L_diag[l_step];
    L_left = &L_left[l_step];
    B_top = B_next;
    B_next = &B_next[b_step];
  }

  if (remain_nx <= _BLOCK_M && remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (L_diag, shm, remain_nx, remain_ny, ld_l, false);
    __syncthreads();

    DenseTrsmL (B_top, shm, nx_b, remain_ny, remain_nx, ld_b, ld_0);
  }
  else if (remain_nx <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (L_diag, shm, remain_nx, _BLOCK_M, ld_l, false);
    __syncthreads();

    DenseTrsmL (B_top, shm, nx_b, _BLOCK_M, remain_nx, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, L_left, B_top, remain_ny - _BLOCK_M, nx_b, remain_nx, ld_b, ld_l, ld_b, false, false, shm);
  }
  else if (remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (L_diag, shm, _BLOCK_M, remain_ny, ld_l, false);
    __syncthreads();

    DenseTrsmL (B_top, shm, nx_b, remain_ny, _BLOCK_M, ld_b, ld_0);
  }

}

/* U is ny_u x nx_u upper triangular and not unit diagonal, B is ny_b by nx_u, solves X x U = B, overwrites X in B. */
DEVICE void blockDenseTrsmR (real_t * __restrict__ B, const real_t * __restrict__ U, const int nx_b, const int ny_b, const int ny_u, const int ld_b, const int ld_u, real_t * __restrict__ shm)
{
  const int u_step = _BLOCK_M * ld_u + _BLOCK_M, b_step = _BLOCK_M; int remain_nx = nx_b, remain_ny = ny_u;
  const real_t * U_diag = U, * U_top = &U[_BLOCK_M];
  real_t * B_left = B, * B_next = &B[b_step];

  while (remain_nx > _BLOCK_M && remain_ny > _BLOCK_M)
  {
    remain_nx -= _BLOCK_M;
    remain_ny -= _BLOCK_M;

    const int ld_0 = matrixCopy_keepT (U_diag, shm, _BLOCK_M, _BLOCK_M, ld_u, false);
    __syncthreads();

    DenseTrsmR (B_left, shm, _BLOCK_M, ny_b, _BLOCK_M, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, B_left, U_top, ny_b, remain_nx, _BLOCK_M, ld_b, ld_b, ld_u, false, false, shm);

    U_diag = &U_diag[u_step];
    U_top = &U_top[u_step];
    B_left = B_next;
    B_next = &B_next[b_step];
  }

  if (remain_nx <= _BLOCK_M && remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, remain_nx, remain_ny, ld_u, false);
    __syncthreads();

    DenseTrsmR (B_left, shm, remain_nx, ny_b, remain_ny, ld_b, ld_0);
  }
  else if (remain_nx <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, remain_nx, _BLOCK_M, ld_u, false);
    __syncthreads();

    DenseTrsmR (B_left, shm, remain_nx, ny_b, _BLOCK_M, ld_b, ld_0);
  }
  else if (remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, _BLOCK_M, remain_ny, ld_u, false);
    __syncthreads();

    DenseTrsmR (B_left, shm, _BLOCK_M, ny_b, remain_ny, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, B_left, U_top, ny_b, remain_nx - _BLOCK_M, remain_ny, ld_b, ld_b, ld_u, false, false, shm);
  }

}

/* U is ny_u x nx_u upper triangular and not unit diagonal, B is ny_b by nx_u, solves X x U = B, overwrites X in B. */
DEVICE void blockDenseTrsmR_transposeB (real_t * __restrict__ B, const real_t * __restrict__ U, const int nx_b, const int ny_b, const int ny_u, const int ld_b, const int ld_u, real_t * __restrict__ shm)
{
  const int u_step = _BLOCK_M * ld_u + _BLOCK_M, b_step = _BLOCK_M * ld_b; int remain_nx = nx_b, remain_ny = ny_u;
  const real_t * U_diag = U, * U_top = &U[_BLOCK_M];
  real_t * B_left = B, * B_next = &B[b_step];

  while (remain_nx > _BLOCK_M && remain_ny > _BLOCK_M)
  {
    remain_nx -= _BLOCK_M;
    remain_ny -= _BLOCK_M;

    const int ld_0 = matrixCopy_keepT (U_diag, shm, _BLOCK_M, _BLOCK_M, ld_u, false);
    __syncthreads();

    DenseTrsmR_transposeB (B_left, shm, _BLOCK_M, ny_b, _BLOCK_M, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, U_top, B_left, remain_nx, ny_b, _BLOCK_M, ld_b, ld_u, ld_b, true, false, shm);

    U_diag = &U_diag[u_step];
    U_top = &U_top[u_step];
    B_left = B_next;
    B_next = &B_next[b_step];
  }

  if (remain_nx <= _BLOCK_M && remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, remain_nx, remain_ny, ld_u, false);
    __syncthreads();

    DenseTrsmR_transposeB (B_left, shm, remain_nx, ny_b, remain_ny, ld_b, ld_0);
  }
  else if (remain_nx <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, remain_nx, _BLOCK_M, ld_u, false);
    __syncthreads();

    DenseTrsmR_transposeB (B_left, shm, remain_nx, ny_b, _BLOCK_M, ld_b, ld_0);
  }
  else if (remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (U_diag, shm, _BLOCK_M, remain_ny, ld_u, false);
    __syncthreads();

    DenseTrsmR_transposeB (B_left, shm, _BLOCK_M, ny_b, remain_ny, ld_b, ld_0);

    blockDenseGemm (-1., 1., B_next, U_top, B_left, remain_nx - _BLOCK_M, ny_b, remain_ny, ld_b, ld_u, ld_b, true, false, shm);
  }

}

/* LU decomposition of matrix of ny by nx, utilizes L1 cache. */
DEVICE void blockDenseGetrf (real_t * __restrict__ M, const int nx, const int ny, const int ld, real_t * __restrict__ shm)
{
  const int iter_step = _BLOCK_M * ld + _BLOCK_M; int remain_nx = nx, remain_ny = ny;
  real_t * M_diag = M, * M_top = &M[_BLOCK_M], * M_left = &M[iter_step - _BLOCK_M], * M_next = &M[iter_step];

  while (remain_nx > _BLOCK_M && remain_ny > _BLOCK_M)
  {
    remain_nx -= _BLOCK_M;
    remain_ny -= _BLOCK_M;

    const int ld_0 = matrixCopy_keepT (M_diag, shm, _BLOCK_M, _BLOCK_M, ld, false);
    __syncthreads();

    DenseGetrf (shm, _BLOCK_M, _BLOCK_M, ld_0);

    matrixCopy (shm, M_diag, _BLOCK_M, _BLOCK_M, ld_0, ld);

    DenseTrsmL (M_top, shm, remain_nx, _BLOCK_M, _BLOCK_M, ld, ld_0);

    DenseTrsmR (M_left, shm, _BLOCK_M, remain_ny, _BLOCK_M, ld, ld_0);

    blockDenseGemm (-1., 1., M_next, M_left, M_top, remain_ny, remain_nx, _BLOCK_M, ld, ld, ld, false, false, shm);

    M_diag = M_next;
    M_left = &M_left[iter_step];
    M_top = &M_top[iter_step];
    M_next = &M_next[iter_step];
  }

  if (remain_nx <= _BLOCK_M && remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (M_diag, shm, remain_nx, remain_ny, ld, false);
    __syncthreads();

    DenseGetrf (shm, remain_nx, remain_ny, ld_0);

    matrixCopy (shm, M_diag, remain_nx, remain_ny, ld_0, ld);
    __syncthreads();
  }
  else if (remain_ny <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (M_diag, shm, _BLOCK_M, remain_ny, ld, false);
    __syncthreads();

    DenseGetrf (shm, _BLOCK_M, remain_ny, ld_0);

    matrixCopy (shm, M_diag, _BLOCK_M, remain_ny, ld_0, ld);

    DenseTrsmL (M_top, shm, remain_nx - _BLOCK_M, remain_ny, _BLOCK_M, ld, ld_0);
  }
  else if (remain_nx <= _BLOCK_M)
  {
    const int ld_0 = matrixCopy_keepT (M_diag, shm, remain_nx, _BLOCK_M, ld, false);
    __syncthreads();

    DenseGetrf (shm, remain_nx, _BLOCK_M, ld_0);

    matrixCopy (shm, M_diag, remain_nx, _BLOCK_M, ld_0, ld);

    DenseTrsmR (M_left, shm, remain_nx, remain_ny - _BLOCK_M, _BLOCK_M, ld, ld_0);
  }

}


/* General Matrix multiplication with 3 matrices. M (m by n) = alpha * A (m by k) * B (k by l) * C (l by n) + beta * old_M. */
DEVICE void blockDenseGemm_3x (const real_t alpha, const real_t beta, real_t * __restrict__ M, const real_t * __restrict__ A, const real_t * __restrict__ B, 
  const real_t * __restrict__ C, const int m, const int n, const int k, const int l, const int ld_m, const int ld_a, const int ld_b, const int ld_c, 
  const bool a_T, const bool b_T, const bool c_T, const int control, real_t * __restrict__ shm, real_t * __restrict__ my_tmp)
{

  real_t * t1 = my_tmp;

  if (control) // (A x B) x C
  {
    blockDenseGemm (1., 0., t1, A, B, m, l, k, l, ld_a, ld_b, a_T, b_T, shm);
    blockDenseGemm (alpha, beta, M, t1, C, m, n, l, ld_m, l, ld_c, false, c_T, shm);
  }
  else // A x (B x C)
  {
    blockDenseGemm (1., 0., t1, B, C, k, n, l, n, ld_b, ld_c, b_T, c_T, shm);
    blockDenseGemm (alpha, beta, M, A, t1, m, n, k, ld_m, ld_a, n, a_T, false, shm);
  }

}

/* General Matrix multiplication with 4 matrices. M (m by n) = alpha * A (m by k) * B (k by l) * C (l by o) * D (o by n) + beta * old_M. */
DEVICE void blockDenseGemm_4x (const real_t alpha, const real_t beta, real_t * __restrict__ M, const real_t * __restrict__ A, const real_t * __restrict__ B, const real_t * __restrict__ C, 
  const real_t * __restrict__ D, const int m, const int n, const int k, const int l, const int o, const int ld_m, const int ld_a, const int ld_b, const int ld_c, 
  const int ld_d, const bool a_T, const bool b_T, const bool c_T, const bool d_T, const int control, const int offset, real_t * __restrict__ shm, real_t * __restrict__ my_tmp)
{

  real_t * t1 = my_tmp, * t2 = &my_tmp[offset];

  switch (control) 
  {
  case 0: // ((A x B) x C) x D, t1 m * l, t2 m * o
  {
    blockDenseGemm (1., 0., t1, A, B, m, l, k, l, ld_a, ld_b, a_T, b_T, shm);
    blockDenseGemm (1., 0., t2, t1, C, m, o, l, o, l, ld_c, false, c_T, shm);
    blockDenseGemm (alpha, beta, M, t2, D, m, n, o, ld_m, o, ld_d, false, d_T, shm);
    break;
  }
  case 1: // (A x (B x C)) x D, t1 k * o, t2 m * o
  {
    blockDenseGemm (1., 0., t1, B, C, k, o, l, o, ld_b, ld_c, b_T, c_T, shm);
    blockDenseGemm (1., 0., t2, A, t1, m, o, k, o, ld_a, o, a_T, false, shm);
    blockDenseGemm (alpha, beta, M, t2, D, m, n, o, ld_m, o, ld_d, false, d_T, shm);
    break;
  }
  case 2: // A x ((B x C) x D), t1 k * o, t2 k * n
  {
    blockDenseGemm (1., 0., t1, B, C, k, o, l, o, ld_b, ld_c, b_T, c_T, shm);
    blockDenseGemm (1., 0., t2, t1, D, k, n, o, n, o, ld_d, false, d_T, shm);
    blockDenseGemm (alpha, beta, M, A, t2, m, n, k, ld_m, ld_a, n, a_T, false, shm);
    break;
  }
  case 3: // A x (B x (C x D)), t1 l * n, t2 k * n
  {
    blockDenseGemm (1., 0., t1, C, D, l, n, o, n, ld_c, ld_d, c_T, d_T, shm);
    blockDenseGemm (1., 0., t2, B, t1, k, n, l, n, ld_b, n, b_T, false, shm);
    blockDenseGemm (alpha, beta, M, A, t2, m, n, k, ld_m, ld_a, n, a_T, false, shm);
    break;
  }
  case 4: // (A x B) x (C x D), t1 m * l, t2 l * n
  {
    blockDenseGemm (1., 0., t1, A, B, m, l, k, l, ld_a, ld_b, a_T, b_T, shm);
    blockDenseGemm (1., 0., t2, C, D, l, n, o, n, ld_c, ld_d, c_T, d_T, shm);
    blockDenseGemm (alpha, beta, M, t1, t2, m, n, l, ld_m, l, n, false, false, shm);
    break;
  }
  default:
  { break; }
  }

}

/* Find the index of the largest absolute value element across the warp. Returns lane number [0, 31]. */
DEVICE int warpReduceMax_Index (const real_t max_in)
{
  real_t max = max_in; int max_lane = lane_rank();

  for (int mask = warpSize / 2; mask > 0; mask /= 2)
  {
    const real_t s_max = __shfl_xor_sync (0xffffffff, max, mask, warpSize);
    const int s_lane = __shfl_xor_sync (0xffffffff, max_lane, mask, warpSize);
    if (s_max > max || (s_max == max && s_lane < max_lane))
    { max = s_max; max_lane = s_lane; }
  }

  return max_lane;
}

/* Find the index of the largest absolute value element in matrix[0], matrix[1], ... matrix[n-1]. Returns [0, n-1]. */
DEVICE int blockReduceMax_Index (const real_t * __restrict__ M, const int n, int * __restrict__ shm)
{
  real_t max = 0; int index = 0;
  
  for (int i = thread_rank(); i < n; i += block_dim())
  {
    const real_t value = abs (M[i]);
    if (value > max)
    { max = value; index = i; }
  }

  if (lane_rank() == warpReduceMax_Index(max))
  { shm[warp_rank()] = index; }
  __syncthreads();

  if (num_warps() > 1 && warp_rank() == 0)
  {
    max = 0; index = 0;
    for (int i = lane_rank(); i < num_warps(); i += warpSize)
    {
      const real_t value = abs (M[ shm[i] ]);
      if (value > max)
      { max = value; index = shm[i]; }
    }

    if (lane_rank() == warpReduceMax_Index(max))
    { shm[0] = index; }
  }

  __syncthreads(); 

  return shm[0];
}

/* Exchange row1[0] with row2[0], row1[1] with row2[1], ... row1[n-1] with row2[n-1]. */
DEVICE void blockSwapRows (real_t * __restrict__ row1, real_t * __restrict__ row2, const int n)
{
  for (int i = thread_rank(); i < n; i += block_dim())
  { const real_t t = row1[i]; row1[i] = row2[i]; row2[i] = t; }
}
 
/* Exchange col1[0] with col2[0], col1[1] with col2[1], ... col1[n-1] with col2[n-1]. */
DEVICE void blockSwapColumns (real_t * __restrict__ col1, real_t * __restrict__ col2, const int n, const int ld)
{
  for (int i = thread_rank(); i < n; i += block_dim())
  { const real_t t = col1[i * ld]; col1[i * ld] = col2[i * ld]; col2[i * ld] = t; }
}

/* Using a group of threads to apply pivot the pivot swaps to the matrix. Recover flag retrieves original matrix. Utilizes L1. */
DEVICE void blockApplyPivot (real_t * __restrict__ M, const int * __restrict__ p, const int nx, const int ny, const int ld, const bool recover, 
  real_t * __restrict__ shm, const int shm_size)
{
  const int step_size = shm_size / ny;

  for (int n = 0; n < nx; n += step_size)
  {
    const int cols = (nx - n > step_size) ? step_size : nx - n;
    for (int i = thread_rank(); i < cols * ny; i++)
    {
      const int row = i / cols, col = i - row * cols, target = p[row];
      if (recover)
      { shm[target * cols + col] = M[row * ld + n + col]; }
      else
      { shm[row * cols + col] = M[target * ld + n + col]; }
    }
    __syncthreads();

    matrixCopy_keepT (&shm[0], &M[n], cols, ny, cols, false);
    __syncthreads();
  }
}

/* Set pivot[0] = 0, pivot[1] = 1, ... pivot[n-1] = n-1. */
DEVICE void resetPivot (int *p, const int n)
{
  for (int i = thread_rank(); i < n; i += block_dim())
  { p[i] = i; }
}

DEVICE int warpAllReduceSum (int value)
{
  for (int mask = warpSize / 2; mask > 0; mask /= 2)
  { value += __shfl_xor_sync(0xffffffff, value, mask, warpSize); }
  return value;
}

DEVICE int blockAllReduceSum (int value, int * shm)
{
  value = warpAllReduceSum (value);

  if (lane_rank() == 0)
  { shm[warp_rank()] = value; }
  __syncthreads();

  if (block_dim() > warpSize && warp_rank() == 0)
  {
    value = 0;
    for (int i = lane_rank(); i < num_warps(); i += warpSize)
    { value += shm[i]; }

    value = warpAllReduceSum (value);
    if (lane_rank() == 0)
    { shm[0] = value; }
  }
  __syncthreads();

  return shm[0];
}


DEVICE bool blockSingleSideJacobiSVD (real_t * __restrict__ UxS, real_t * __restrict__ VT, const int nx, const int ny, const int ld_UxS, const int ld_VT, real_t * __restrict__ shm, const real_t epi)
{
  bool iter = false;

  for (int step = nx / 2; step > 0; step /= 2) for (int offset = 0; offset < step; offset++)
  {
    for (int i = thread_rank(); i < 2 * nx; i += block_dim())
    { shm[i] = 0.; }
    __syncthreads();

    for (int row = 0; row < ny; row++)
    {
      const int row_UxS = row * ld_UxS;
      for (int col = thread_rank(); col < nx; col += block_dim())
      {
        const real_t e = UxS[row_UxS + col];
        shm[col] += e * e;
        const int lane = col % (2 * step);
        if (lane < step)
        { 
          if (lane + offset >= step)
          { shm[nx + col] += e * UxS[row_UxS + col + offset]; }
          else
          { shm[nx + col] += e * UxS[row_UxS + col + step + offset]; }
        }
      }
    }
    __syncthreads();

    for (int col = warp_rank(); col < nx; col += num_warps())
    {
      if (shm[nx + col] > epi || shm[nx + col] < -epi)
      {
        double sine, cosine;
        const int col2 = col + offset + ((col % step + offset >= step) ? 0 : step);
        const bool swap = shm[col] < shm[col2];
        iter = true;

        if (lane_rank() == 0)
        {
          const double torque = (shm[col2] - shm[col]) / (shm[nx + col] * 2.);
          const double tangent = (signbit(torque) * -2 + 1) / (fabs(torque) + sqrt(1. + torque * torque));
          cosine = rsqrt(1. + tangent * tangent);
          sine = cosine * tangent;
        }
        __syncwarp();

        cosine = __shfl_sync(0xffffffff, cosine, 0, warpSize);
        sine = __shfl_sync(0xffffffff, sine, 0, warpSize);

        for (int row = lane_rank(); row < ny; row += warpSize)
        {
          const int row_UxS = row * ld_UxS;
          const real_t e1 = swap ? UxS[row_UxS + col2] : UxS[row_UxS + col], e2 = swap ? UxS[row_UxS + col] : UxS[row_UxS + col2];
          UxS[row_UxS + col] = cosine * e1 - sine * e2;
          UxS[row_UxS + col2] = sine * e1 + cosine * e2;
        }

        for (int row = lane_rank(); row < nx; row += warpSize)
        {
          const int row_Vreal_t = row * ld_VT;
          const real_t e3 = swap ? VT[row_Vreal_t + col2] : VT[row_Vreal_t + col], e4 = swap ? VT[row_Vreal_t + col] : VT[row_Vreal_t + col2];
          VT[row_Vreal_t + col] = cosine * e3 - sine * e4;
          VT[row_Vreal_t + col2] = sine * e3 + cosine * e4;
        }
        __syncwarp();

      }
    }
    __syncthreads();
  }

  return iter;
}


DEVICE void blockGivensRotation (real_t * __restrict__ M, const int nx, const int ny, const int ld_m)
{
  const int l_id = lane_rank(), w_id = warp_rank(), n_wp = num_warps(), n = nx + ny - 2;

  for (int iter = 0; iter < n; iter++)
  {
    for (int col = w_id; col < nx; col += n_wp)
    {
      const int row = ny - 2 + 2 * col - iter;
      if (row < ny - 1 && row >= col)
      {
        const int row2 = row + 1;
        real_t cosine, sine, * M_row = &M[row * ld_m], * M_row2 = &M[row2 * ld_m]; 

        if (l_id == 0)
        {
          real_t a = M_row[col], b = M_row2[col], r, p;

          if (b == 0)
          { cosine = copysign(1., a); sine = 0.; r = fabs(a); p = 0.; }
          else if (a == 0)
          { cosine = 0.; sine = copysign(1., b); r = fabs(b); p = 1.; }
          else if (fabs(b) > fabs(a))
          { real_t t = - a / b; sine = rhypot(1., t); cosine = sine * t; r = - b / sine; p = 2. / cosine; } // rhypot(1, t) = 1 / sqrt(1 + t * t);
          else
          { real_t t = - b / a; cosine = rhypot(1., t); sine = cosine * t; r = a / cosine; p = sine / 2.; }

          M_row[col] = r;
          M_row2[col] = p;
        }

        cosine = __shfl_sync(0xffffffff, cosine, 0, warpSize);
        sine = __shfl_sync(0xffffffff, sine, 0, warpSize);

        for (int i = col + l_id + 1; i < nx; i += warpSize)
        {
          const real_t a = M_row[i], b = M_row2[i];
          M_row[i] = fma (cosine, a, fma (- sine, b, 0.)); // cosine * a - sine * b;
          M_row2[i] = fma (sine, a, fma (cosine, b, 0.)); // sine * a + cosine * b;
        }
      }
    }
    __syncthreads();
  }
}

DEVICE void blockGivensRecoverQ (real_t * __restrict__ Q, const real_t * __restrict__ R, const int nx, const int ny, const int p, const int ld_q, const int ld_r)
{
  const int l_id = lane_rank(), w_id = warp_rank(), n_wp = num_warps(), n = nx + ny - 2;

  for (int row = w_id; row < ny; row += n_wp) for (int col = l_id; col < p; col += warpSize)
  { Q[row * ld_q + col] = (real_t) (row == col); }
  __syncthreads();

  for (int iter = 0; iter < n; iter++)
  {
    for (int col = w_id; col < nx; col += n_wp)
    {
      const int row = 2 * col + 1 - nx + iter;
      if (row < ny - 1 && row >= col)
      {
        const int row2 = row + 1;
        real_t cosine, sine, * Q_row = &Q[row * ld_q], * Q_row2 = &Q[row2 * ld_q];

        if (l_id == 0)
        {
          real_t p = R[row2 * ld_r + col];

          if (p == 0)
          { cosine = 1.; sine = 0.; }
          else if (p == 1)
          { cosine = 0.; sine = 1.; }
          else if (fabs(p) > 2)
          { cosine = 2. / p; sine = sqrt (fma (cosine, - cosine, 1.)); }
          else
          { sine = 2. * p; cosine = sqrt (fma (sine, - sine, 1.)); }

        }

        cosine = __shfl_sync(0xffffffff, cosine, 0, warpSize);
        sine = __shfl_sync(0xffffffff, sine, 0, warpSize);

        for (int i = col + l_id; i < p; i += warpSize)
        {
          const real_t a = Q_row[i], b = Q_row2[i];
          Q_row[i] = fma (cosine, a, fma (sine, b, 0.)); // cosine * a + sine * b;
          Q_row2[i] = fma (- sine, a, fma (cosine, b, 0.)); // - sine * a + cosine * b;
        }

      }
    }
    __syncthreads();
  }
}

DEVICE void blockLowRankAccum (real_t * __restrict__ U1, real_t * __restrict__ VT1, const real_t * __restrict__ U2, const real_t * __restrict__ VT2, const int nx, const int ny, const int k1, const int k2, 
  const int ld_u1, const int ld_vt1, const int ld_u2, const int ld_vt2, const int offset1, const int offset2, real_t * __restrict__ shm, real_t * __restrict__ my_tmp, const real_t *__restrict__ rnd_seed)
{
  real_t * U = my_tmp, * V = &my_tmp[offset1], * Q = &my_tmp[offset2];

  blockDenseGemm (1., 0., Q, rnd_seed, VT1, k1, k1, nx, k1, k1, ld_vt1, true, false, shm);
  blockDenseGemm (1., 0., U, U1, Q, ny, k1, k1, k1, ld_u1, k1, false, true, shm);

  blockDenseGemm (1., 0., Q, rnd_seed, VT2, k1, k2, nx, k2, k1, ld_vt2, true, false, shm);
  blockDenseGemm (1., 1., U, U2, Q, ny, k1, k2, k1, ld_u2, k2, false, true, shm);

  blockGivensRotation (U, k1, ny, k1);
  blockGivensRecoverQ (Q, U, k1, ny, k1, k1, k1);

  blockDenseGemm (1., 0., U, Q, U1, k1, k1, ny, k1, k1, ld_u1, true, false, shm);
  blockDenseGemm (1., 0., V, VT1, U, nx, k1, k1, k1, ld_vt1, k1, false, true, shm);

  matrixCopy (Q, U1, k1, ny, k1, ld_u1);

  blockDenseGemm (1., 0., U, Q, U2, k1, k2, ny, k2, k1, ld_u2, true, false, shm);
  blockDenseGemm (1., 1., V, VT2, U, nx, k1, k2, k1, ld_vt2, k2, false, true, shm);

  matrixCopy (V, VT1, k1, nx, k1, ld_vt1);
  __syncthreads();

}

DEVICE int blockReadRank (real_t * __restrict__ A, const int nx, const int ny, const int ld, const double epi, real_t * __restrict__ shm, const int shm_size)
{
  const int step = shm_size / nx, total = step * nx;

  for (int i = thread_rank(); i < total; i += block_dim())
  { shm[i] = 0; }
  __syncthreads();

  for (int i = 0; i < ny; i += step)
  {
    for (int j = thread_rank(); j < total; j += block_dim())
    {
      const int row = i + j / nx, col = j - (row - i) * nx;
      const real_t e = A[row * ld + col];
      shm[j] += e * e;
    }
  }
  __syncthreads();

  int r = 0;
  for (int i = thread_rank(); i < nx; i += block_dim())
  {
    real_t norm = 0;
    for (int j = 0; j < step; j++)
    { norm += shm[j * nx + i]; }
    r += (int) (norm >= epi);
  }
  __syncthreads();
  
  const int r_ = blockAllReduceSum (r, (int *) shm);
  __syncthreads();

  return r_;
}

DEVICE int blockRandomizedSVD (real_t * __restrict__ A, real_t * __restrict__ VT, const int nx, const int ny, const int ld_a, const int ld_v, 
  const int rank, const double epi, const int iter_limit, real_t * __restrict__ shm, const int shm_size)
{
  /*const int P = rank > nx ? (nx > ny ? ny : nx) : (rank > ny ? ny : rank);

  real_t * X, ** X_ptr = (real_t **) &shm[0], *Y, **Y_ptr = (real_t **) &shm[1], *B, ** B_ptr = (real_t **) &shm[2];
  if (thread_rank() == 0)
  { 
    X = new T[ny * P]; *X_ptr = X; 
    Y = new T[ny * P]; *Y_ptr = Y; 
    B = new T[P * nx]; *B_ptr = B; 
  }
  __syncthreads();

  X = *X_ptr; Y = *Y_ptr; B = *B_ptr;
  __syncthreads();

  blockDenseGemm_shm (1., 0., X, A, dev_rnd_seed, ny, P, nx, P, ld_a, P, false, false, shm, shm_size);

  matrixCopy_fromRM (X, Y, P, ny, P, P, false);
  blockGivensRotation (X, P, ny, P);
  blockDenseTrsmR_shm (Y, X, P, ny, P, P, P, false, shm, shm_size);
  blockGramSchmidt (Y, P, ny, P, shm);

  blockDenseGemm_shm (1., 0., B, Y, A, P, nx, ny, nx, P, ld_a, true, false, shm, shm_size);*/

  int * iter = (int *) &shm[0], *loop_counter = (int *) &shm[1];
  if (thread_rank() == 0)
  { *iter = 1; *loop_counter = 0; }
  __syncthreads();

  while (*iter && *loop_counter < iter_limit)
  {
    if (thread_rank() == 0)
    { *iter = 0; (*loop_counter)++; }
    __syncthreads();

    bool iter_result = blockSingleSideJacobiSVD(A, VT, nx, ny, ld_a, ld_v, &shm[2], epi);
    if (thread_rank() == 0)
    { *iter = (int) iter_result; }
    __syncthreads();
  }

  /*blockDenseGemm_shm (1., 0., A, Y, B, ny, nx, P, ld_a, P, nx, false, false, shm, shm_size);
  const int r = blockReadRank <T> (B, nx, P, nx, epi, shm, shm_size);
  __syncthreads();

  if (thread_rank() == 0)
  { delete X; delete Y; delete B; }
  __syncthreads();*/

  const int r = blockReadRank (A, nx, ny, nx, epi, shm, shm_size);

  return r;

}


__global__ void kernel_dynamic (const int ** __restrict__ insts, void ** __restrict__ ptrs, volatile int * __restrict__ comm_space, 
  real_t ** __restrict__ block_tmps, real_t * __restrict__ dev_rnd_seed, unsigned long long ** __restrict__ clocks)
{
  __shared__ int shm [_SHM_SIZE]; 

  const int * pc = insts [block_rank()], t_id = thread_rank(); 
  real_t * my_tmp = block_tmps[block_rank()];
  unsigned long long * clocks_block = clocks[block_rank()];

load_inst:
  int next_pc = 0;
  const int * signal_id = nullptr;
  if (t_id < _MAX_INST_LENGTH)
  { shm[t_id] = pc[t_id]; }
  if (t_id == 0)
  { clocks_block[0] = clock64(); clocks_block = &clocks_block[1]; }
  __syncthreads();

  switch ((opcode_t) shm[0])
  {
  case execute: 
  { signal_id = &pc[1]; goto exe; }
  case signal_wait: 
  { goto wait; }
  case finish: default: 
  { goto fin; }
  }

exe:
  switch ((operation_t) shm[2])
  {
  case nop:
  { next_pc = nop_l; goto write; }
  case getrf:
  {
    real_t * M = (real_t *) ptrs[shm[3]]; 
    const int offset = shm[4], nx = shm[5], ny = shm[6], ld = shm[7];
    __syncthreads();
    blockDenseGetrf (&M[offset], nx, ny, ld, (real_t *) shm);
    next_pc = getrf_l; goto write;  
  }

  case trsml:
  {
    real_t * B = (real_t *) ptrs[shm[3]], * L = (real_t *) ptrs[shm[4]];
    const int offset_b = shm[5], offset_l = shm[6], nx_b = shm[7], ny_b = shm[8], nx_l = shm[9], ld_b = shm[10], ld_l = shm[11];
    const bool b_T = (bool) shm[12];
    __syncthreads();
    if (b_T)
    { }
    else
    { blockDenseTrsmL (&B[offset_b], &L[offset_l], nx_b, ny_b, nx_l, ld_b, ld_l, (real_t *) shm); }
    next_pc = trsml_l; goto write;
  }

  case trsmr:
  {
    real_t * B = (real_t *) ptrs[shm[3]], * U = (real_t *) ptrs[shm[4]];
    const int offset_b = shm[5], offset_u = shm[6], nx_b = shm[7], ny_b = shm[8], ny_u = shm[9], ld_b = shm[10], ld_u = shm[11];
    const bool b_T = (bool) shm[12];
    __syncthreads();
    if (b_T)
    { blockDenseTrsmR_transposeB (&B[offset_b], &U[offset_u], nx_b, ny_b, ny_u, ld_b, ld_u, (real_t *) shm); }
    else
    { blockDenseTrsmR (&B[offset_b], &U[offset_u], nx_b, ny_b, ny_u, ld_b, ld_u, (real_t *) shm); }
    next_pc = trsmr_l; goto write;
  }

  case gemm:
  {
    real_t * M = (real_t *) ptrs[shm[3]], * A = (real_t *) ptrs[shm[4]], * B = (real_t *) ptrs[shm[5]];
    const int offset_m = shm[6], offset_a = shm[7], offset_b = shm[8], m = shm[9], n = shm[10], k = shm[11], ld_m = shm[12], ld_a = shm[13], ld_b = shm[14];
    const bool a_T = (bool) shm[15], b_T = (bool) shm[16];
    __syncthreads();
    blockDenseGemm (-1., 1., &M[offset_m], &A[offset_a], &B[offset_b], m, n, k, ld_m, ld_a, ld_b, a_T, b_T, (real_t *) shm);
    next_pc = gemm_l; goto write;
  }

  case gemm_plus:
  {
    real_t * M = (real_t *) ptrs[shm[3]], * A = (real_t *) ptrs[shm[4]], * B = (real_t *) ptrs[shm[5]];
    const int offset_m = shm[6], offset_a = shm[7], offset_b = shm[8], m = shm[9], n = shm[10], k = shm[11], ld_m = shm[12], ld_a = shm[13], ld_b = shm[14];
    const bool a_T = (bool) shm[15], b_T = (bool) shm[16];
    __syncthreads();
    blockDenseGemm (1., 1., &M[offset_m], &A[offset_a], &B[offset_b], m, n, k, ld_m, ld_a, ld_b, a_T, b_T, (real_t *) shm);
    next_pc = gemm_plus_l; goto write;
  }

  case gemm_3x:
  {
    real_t * M = (real_t *) ptrs[shm[3]], * A = (real_t *) ptrs[shm[4]], * B = (real_t *) ptrs[shm[5]], * C = (real_t *) ptrs[shm[6]];
    const int offset_m = shm[7], offset_a = shm[8], offset_b = shm[9], offset_c = shm[10], m = shm[11], n = shm[12], k = shm[13], l = shm[14];
    const int ld_m = shm[15], ld_a = shm[16], ld_b = shm[17], ld_c = shm[18];
    const bool a_T = (bool) shm[19], b_T = (bool) shm[20], c_T = (bool) shm[21];
    const int control = shm[22];
    __syncthreads();
    blockDenseGemm_3x (-1., 1., &M[offset_m], &A[offset_a], &B[offset_b], &C[offset_c], m, n, k, l, ld_m, ld_a, ld_b, ld_c, a_T, b_T, c_T, control, (real_t *) shm, my_tmp);
    next_pc = gemm_3x_l; goto write;
  }

  case gemm_4x:
  {
    real_t * M = (real_t *) ptrs[shm[3]], * A = (real_t *) ptrs[shm[4]], * B = (real_t *) ptrs[shm[5]], * C = (real_t *) ptrs[shm[6]], * D = (real_t *) ptrs[shm[7]];
    const int offset_m = shm[8], offset_a = shm[9], offset_b = shm[10], offset_c = shm[11], offset_d = shm[12];
    const int m = shm[13], n = shm[14], k = shm[15], l = shm[16], o = shm[17];
    const int ld_m = shm[18], ld_a = shm[19], ld_b = shm[20], ld_c = shm[21], ld_d = shm[22];
    const bool a_T = (bool) shm[23], b_T = (bool) shm[24], c_T = (bool) shm[25], d_T = (bool) shm[26];
    const int control = shm[27], offset = shm[28];
    __syncthreads();
    blockDenseGemm_4x (-1., 1., &M[offset_m], &A[offset_a], &B[offset_b], &C[offset_c], &D[offset_d], m, n, k, l, o, ld_m, ld_a, ld_b, ld_c, ld_d, a_T, b_T, c_T, d_T, control, offset, (real_t *) shm, my_tmp);

    next_pc = gemm_4x_l; goto write;
  }

  case accum:
  {
    real_t * U1 = (real_t *) ptrs[shm[3]], * VT1 = (real_t *) ptrs[shm[4]], * U2 = (real_t *) ptrs[shm[5]], * VT2 = (real_t *) ptrs[shm[6]];
    const int offset_u1 = shm[7], offset_vt1 = shm[8], offset_u2 = shm[9], offset_vt2 = shm[10];
    const int nx = shm[11], ny = shm[12], rank1 = shm[13], rank2 = shm[14], ld_u1 = shm[15], ld_vt1 = shm[16], ld_u2 = shm[17], ld_vt2 = shm[18];
    const int offset1 = shm[19], offset2 = shm[20];
    __syncthreads();
    blockLowRankAccum (&U1[offset_u1], &VT1[offset_vt1], &U2[offset_u2], &VT2[offset_vt2], nx, ny, rank1, rank2, ld_u1, ld_vt1, ld_u2, ld_vt2, offset1, offset2, (real_t *) shm, my_tmp, dev_rnd_seed);
    next_pc = accum_l; goto write;
  }

  default: goto fin;
  }

wait:
  if (t_id == 0)
  { wait(shm[2]); shm[0] = comm_space[shm[1]]; }
  __syncthreads();
  if (shm[0])
  { next_pc = 3; }
  goto sync;

write:
  if (t_id == 0)
  { comm_space[* signal_id] = 1; }
  __threadfence();
  goto sync;

sync:
  __syncthreads();
  if (next_pc > 0) 
  { pc = &pc[next_pc]; goto load_inst; }
  else
  { goto wait; }
  
fin:
  return;
}

