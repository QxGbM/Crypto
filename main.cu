#include "hip/hip_runtime.h"

#include <pspl.cuh>

__global__ void test_kernel(inst_handler <float> ih) { ih.run(); }

__host__ int test0()
{
  dev_hierarchical <double> *a = new dev_hierarchical <double>(2, 2);
  a->loadTestMatrix(2, 2, 4);
  a->print();

  multi_level_index *id = new multi_level_index();
  ops_chain *ops = get_ops_h_getrf(a, id);
  ops->print();

  dag *d = new dag(ops);

  d->print();

  timer *myTimer = new timer();
  myTimer->newEvent("TEST", start);

  void ** args = d -> getArgsAddress();
  hipLaunchKernel((void *)kernel_dynamic, 4, 256, args);
  delete[] args;

  myTimer->newEvent("TEST", end);
  myTimer->printStatus();
  myTimer->dumpAllEvents_Sync();
  hipDeviceSynchronize();

  delete a;
  delete ops;
  delete id;
  delete d;
  delete myTimer;

  hipDeviceReset();

  return 0;
}

__host__ int test1 ()
{
  hipSetDevice(0);

  dev_dense <float> *a = new dev_dense <float> (16, 16, 0, false);
  a -> loadTestMatrix();

  hipStream_t main_stream;
  hipStreamCreate(&main_stream);

  inst_handler <float> *ih = new inst_handler <float> (5);
  ih -> set_getrf_inst(0, &(a -> getElements())[0], 8, 8, 16);
  ih -> set_trsml_inst(1, &(a -> getElements())[0], &(a -> getElements())[8], 8, 8, 8, 16, 16);
  ih -> set_trsmr_inst(2, &(a -> getElements())[0], &(a -> getElements())[128], 8, 8, 8, 16, 16);
  ih -> set_gemm_inst(3, &(a -> getElements())[136], &(a -> getElements())[128], &(a -> getElements())[8], 8, 8, 8, 16, 16, 16);
  ih -> set_getrf_inst(4, &(a -> getElements())[136], 8, 8, 16);

  ih -> add_dep(0, 1);
  ih -> add_dep(0, 2);
  ih -> add_dep(1, 3);
  ih -> add_dep(2, 3);
  ih -> add_dep(3, 4);

  ih -> print();

  timer myTimer = timer();

  myTimer.newEvent("GETRF", start, main_stream);
  hipLaunchKernel((void *)test_kernel, 4, 1024, (void **) &ih, 0, main_stream);
  myTimer.newEvent("GETRF", end, main_stream);

  myTimer.printStatus();
  myTimer.dumpAllEvents_Sync();

  dev_dense <float> *b = a -> restoreLU();
  a -> loadTestMatrix();
  printf("Rel. L2 Error: %e\n\n", b -> L2Error(a));

  delete a;
  delete b;
  hipStreamDestroy(main_stream);
  hipDeviceReset();

  return 0;
}


int main(int argc, char **argv)
{
  test1();
  //test0();

  return 0;
}