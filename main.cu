#include "hip/hip_runtime.h"

#include <pspl.cuh>
#define ref


__global__ void partial_pivot_kernel(double *matrix, const int nx, const int ny, const int ld, int *pivot)
{
  __shared__ double shm[6144];
  blockDenseGetrf_shm <double>(matrix, pivot, nx, ny, ld, &shm[0]);
}

__global__ void recover_pivot_kernel(double *matrix, const int nx, const int ny, const int ld, int *pivot)
{
  __shared__ double shm[6144];
  blockApplyPivot <double>(matrix, pivot, nx, ny, ld, true, &shm[0], 6144);
}

template <class T> __host__ int test0()
{
  hipSetDevice(0);
  hipDeviceReset();

  const int n = 2, levels = 1, dim = 4, admis = 3;

  dev_hierarchical <T> *a = new dev_hierarchical <T> (n, n);
  a -> loadTestMatrix(levels, n, dim, admis);

  const int blocks = 160, threads = 1024;

#ifdef ref
  dev_dense <T> *c = a -> convertToDense();
  printf("Reference Matrix converted to dense.\n");
#endif // ref

  hipError_t error = hierarchical_GETRF <T, 12288> (a, blocks, threads);

#ifdef ref
  if (error == hipSuccess)
  {
    dev_dense <T> *b = a -> convertToDense();
    partial_pivot_kernel <<<1, 1024, 0, 0 >>> (c -> getElements(), a -> getNx_abs(), a -> getNy_abs(), a -> getNx_abs(), nullptr);
    hipDeviceSynchronize();

    printf("Rel. L2 Error: %e\n\n", b -> L2Error(c));
    delete b;
  }
  delete c;
#endif // ref

  delete a;

  return 0;
}

__global__ void svd_kernel (double * U, double * VT, const int nx, const int ny, const int ld_u, const int ld_v)
{
  __shared__ double shm[6144];
  int i = blockRandomizedSVD <double> (U, VT, nx, ny, ld_u, ld_v, 8, 1.0e-14, 100, shm, 6144);
  if (thread_rank() == 0) { printf("iters: %d\n", i); }
}

__global__ void qr_kernel (double* Q, double* R, const int nx, const int ny, const int ld_q, const int ld_r)
{
  __shared__ double shm[6144];
  matrixCopy_fromRM (R, Q, nx, ny, ld_r, ld_q, false);
  blockGivensRotation (R, nx, ny, ld_r);
  blockDenseTrsmR_shm (Q, R, nx, ny, nx, ld_q, ld_r, false, shm, 6144);

}

int test1()
{
  hipSetDevice(0);
  hipDeviceReset();

  const int nx = 4, ny = 4;

  srand(200);
  double * rnd_seed = new double[_RND_SEED_LENGTH];
#pragma omp parallel for
  for (int i = 0; i < _RND_SEED_LENGTH; i++) { rnd_seed[i] = (double) rand() / RAND_MAX; }

  hipMemcpyToSymbol(HIP_SYMBOL(dev_rnd_seed), rnd_seed, _RND_SEED_LENGTH * sizeof(double), 0, hipMemcpyHostToDevice);

  dev_low_rank <double> *A = new dev_low_rank <double> (nx, ny);

  //A -> getUxS() -> loadTestMatrix(20);
  //A -> getVT() -> loadIdentityMatrix();
  A->getVT()->loadTestMatrix();

  timer myTimer = timer();

  myTimer.newEvent("SVD", start);
  //svd_kernel <<<1, 1024>>> (A -> getUxS() -> getElements(), A -> getVT() -> getElements(), nx, ny, nx, nx);
  qr_kernel <<<1, 1024 >>> (A->getUxS()->getElements(), A->getVT()->getElements(), nx, ny, nx, nx);
  myTimer.newEvent("SVD", end);

  myTimer.dumpAllEvents_Sync();
  A->getUxS()->print();
  A->getVT()->print();

  dev_dense <double> *b = A->getUxS()->matrixMultiplication(A->getVT()), *c = new dev_dense<double>(nx, ny);
  c->loadTestMatrix();
  printf("Rel. L2 Error: %e\n\n", c->L2Error(b));
  dev_dense <double>* d = A->getUxS()->matrixMultiplication(A->getUxS()->transpose());
  d->print();

  delete A; delete b; delete c;


  return 0;
}



__host__ int test2()
{
  hipSetDevice(0);
  hipDeviceReset();
  const int nx = 512, ny = 512;

  dev_dense <double> *a = new dev_dense <double> (nx, ny, nx, true);
  a -> loadRandomMatrix(-10, 10, 999);

  timer myTimer = timer();

  myTimer.newEvent("pivot", start);
  partial_pivot_kernel <<<1, 1024, 0, 0 >>> (a -> getElements(), nx, ny, nx, a -> getPivot());
  myTimer.newEvent("pivot", end);
  hipDeviceSynchronize();

  dev_dense <double> *b = a -> restoreLU();

  myTimer.newEvent("pivot recovery", start);
  recover_pivot_kernel <<<1, 1024, 0, 0 >>> (b -> getElements(), nx, ny, nx, a->getPivot());
  myTimer.newEvent("pivot recovery", end);

  myTimer.printStatus();
  myTimer.dumpAllEvents_Sync();

  a->loadRandomMatrix(-10, 10, 999);
  printf("Rel. L2 Error: %e\n\n", b -> L2Error(a));

  delete a;
  delete b;

  return 0;
}

void test3()
{
  
}


int main(int argc, char **argv)
{
  //test0 <double> ();
  test1();
  //test2();

  return 0;
}