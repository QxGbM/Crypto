#include "hip/hip_runtime.h"

#include <dag.cuh>
#include <kernel.cuh>
#include <timer.cuh>
#include <dev_dense_funcs.cuh>
#include <pivot.cuh>
#include <inst_handler.cuh>

__host__ int test0()
{
  dev_hierarchical <double> *a = new dev_hierarchical <double>(2, 2);
  a->loadTestMatrix(2, 2, 4);
  a->print();

  multi_level_index *id = new multi_level_index();
  ops_chain *ops = get_ops_h_getrf(a, id);
  ops->print();

  dag *d = new dag(ops);

  d->print();

  timer *myTimer = new timer();
  myTimer->newEvent("TEST", start);

  void ** args = d -> getArgsAddress();
  hipLaunchKernel((void *)kernel_dynamic, 4, 256, args);
  delete[] args;

  myTimer->newEvent("TEST", end);
  myTimer->printStatus();
  myTimer->dumpAllEvents_Sync();
  hipDeviceSynchronize();

  delete a;
  delete ops;
  delete id;
  delete d;
  delete myTimer;

  hipDeviceReset();

  return 0;
}

__global__ void dense_getrf_kernel(double *matrix, const int nx, const int ny, const int ld)
{
  blockDenseGetrf <double>(matrix, nx, ny, ld);
}

__host__ int test1 ()
{
  const int x = 512, y = 512;

  hipSetDevice(0);

  dev_dense <double> *a = new dev_dense <double> (x, y, 1024);
  a->loadTestMatrix();
  int *dim = a -> getDim3(), nx = dim[0], ny = dim[1], ld = dim[2];
  double *matrix = a -> getElements();
  delete[] dim;

  hipStream_t main_stream;
  hipStreamCreate(&main_stream);

  timer myTimer = timer();
  void ** args = new void *[4]{ &matrix, &nx, &ny, &ld };

  myTimer.newEvent("GETRF", start, main_stream);
  hipLaunchKernel((void *)dense_getrf_kernel, 1, 1024, args);

  myTimer.newEvent("GETRF", end, main_stream);
  hipStreamDestroy(main_stream);

  myTimer.printStatus();
  myTimer.dumpAllEvents_Sync();

  printf("Cuda Execution: getrf finished.\n\n");

  dev_dense <double> *b = a -> restoreLU();
  a -> loadTestMatrix();

  printf("Rel. L2 Error: %e\n\n", b -> L2Error(a));
  printf("-------- n x n Dense GETRF test finished --------\n\n");

  delete a;
  delete b;

  hipDeviceReset();

  return 0;
}

__global__ void pivot_getrf_kernel(int *pivot, double *matrix, const int nx, const int ny, const int ld)
{
  blockDenseGetrf <double> (matrix, nx, ny, ld, pivot);
}

__global__ void recover_pivot_kernel(int *pivot, double *matrix, const int nx, const int ny, const int ld)
{
  blockApplyPivot <double> (matrix, pivot, nx, ny, ld, true);
}

__host__ int test2()
{
  hipSetDevice(0);
  const int x = 512, y = 512;

  dev_dense <double> *a = new dev_dense <double>(x, y, 1024);
  a -> loadRandomMatrix(-10, 10, 999);

  int *dim = a -> getDim3(), nx = dim[0], ny = dim[1], ld = dim[2];
  double *matrix = a -> getElements();
  int *pivot = a -> getPivot();
  delete[] dim;

  timer myTimer = timer();
  void ** args = new void *[5]{ &pivot, &matrix, &nx, &ny, &ld };

  myTimer.newEvent("PIVOT GETRF", start);
  hipLaunchKernel((void *)pivot_getrf_kernel, 1, 1024, args);
  myTimer.newEvent("PIVOT GETRF", end);
  hipDeviceSynchronize();
  delete[] args;

  dev_dense <double> *b = a->restoreLU();
  double *matrix_b = b->getElements();
  void ** args2 = new void *[5]{ &pivot, &matrix_b, &nx, &ny, &nx };

  myTimer.newEvent("PIVOT", start);
  hipLaunchKernel((void *)recover_pivot_kernel, 1, 1024, args2);
  myTimer.newEvent("PIVOT", end);

  myTimer.printStatus();
  myTimer.dumpAllEvents_Sync();
  delete[] args2;

  a->loadRandomMatrix(-10, 10, 999);
  printf("Rel. L2 Error: %e\n\n", b->L2Error(a));

  hipDeviceReset();

  delete a;
  delete b;

  return 0;
}

__global__ void test_kernel(inst_handler <double> ih)
{
  ih.func();
}

__host__ int test3()
{
  hipSetDevice(0);
  inst_handler <double> ih = inst_handler <double> (4);
  ih.change_ptrs_size(32);
  double *a = new double[16];
  double *b = new double[16];
  ih.set_getrf_inst(0, a, 4, 4, 4);
  ih.set_getrf_inst(1, b, 5, 6, 7);
  ih.set_getrf_inst(2, a, 8, 9, 10);
  ih.print();
  test_kernel <<<1, 32>>> (ih);
  hipDeviceReset();
  return 0;
}

int main(int argc, char **argv)
{
  test3();
  //test2();
  //test1();
  //test0();

  return 0;
}