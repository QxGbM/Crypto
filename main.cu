
#define RUN
#ifdef RUN

#include <matrix/Dense.cuh>
#include <matrix/Hierarchical.cuh>
#include <launcher.cuh>
#include <timer.cuh>
#include <hipsolver.h>


int test0 (char test_name[], const int blocks, const int threads, const int kernel_size, const bool ref, char ref_name[])
{
  hipSetDevice(0);
  hipDeviceReset();

  Hierarchical * a = Hierarchical :: readFromFile(test_name, 0);
  //a->print();

  hipError_t error = dev_hierarchical_GETRF(a, blocks, threads, kernel_size);

  if (ref && error == hipSuccess)
  {
    Dense * b = a->convertToDense(), * c = Dense :: readFromFile(ref_name, 0);
    b->print();
    c->print();

    int m = c->getNy();
    int n = c->getNx();
    int ld = c->getLd();
    auto arr = c->getElements();

    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);
    double* Workspace;
    int Lwork;
    int* devInfo;
    hipsolverDnDgetrf_bufferSize(handle, m, n, arr, ld, &Lwork);
    Lwork = 16384;
    hipMalloc(&Workspace, Lwork);
    hipMalloc(&devInfo, sizeof(int));

    timer my_timer = timer();
    my_timer.newEvent("ref", start);
    hipsolverDnDgetrf(handle, m, n, arr, ld, Workspace, nullptr, devInfo);
    my_timer.newEvent("ref", end);


    my_timer.dumpAllEvents_Sync();

    printf("\033[0;31m");
    printf("Rel. L2 Error: %e\n\n", c -> L2Error(b)); 
    printf("\033[0m");

    delete b; b = nullptr;
    delete c; c = nullptr;
  }

  delete a;

  return 0;
}



int main(int argc, char * argv[])
{
  int blocks = 80, threads = 512, kernel_size = 0, rank = _SHADOW_RANK;
  bool ref = false;

  char tmp[32], dir[32] = "bin/", ref_name[32], test_name[32] = "bin/test";

  for (int i = 1; i < argc; i++)
  {
    if (strncmp(argv[i], "-blocks=", 8) == 0)
    { sscanf(argv[i], "-blocks=%d", &blocks); }
    else if (strncmp(argv[i], "-threads=", 9) == 0)
    { sscanf(argv[i], "-threads=%d", &threads); }
    else if (strncmp(argv[i], "-kernel=", 8) == 0)
    { sscanf(argv[i], "-kernel=%d", &kernel_size); }
    else if (strncmp(argv[i], "-rank=", 6) == 0)
    { sscanf(argv[i], "-rank=%d", &rank); }
    else if (strncmp(argv[i], "-dir=", 5) == 0)
    { sscanf(argv[i], "-dir=%s", dir); }
    else if (strncmp(argv[i], "-test=", 6) == 0)
    { sscanf(argv[i], "-test=%s", tmp); strcpy(test_name, dir); strcat(test_name, tmp); }
    else if (strncmp(argv[i], "-ref=", 5) == 0)
    { sscanf(argv[i], "-ref=%s", tmp); strcpy(ref_name, dir); strcat(ref_name, tmp); ref = true; }
    else if (strcmp(argv[i], "-noref") == 0)
    { ref = false; }
    else
    { printf("Unrecognized Arg: %s.\n", argv[i]); }
  }

  test0 (test_name, blocks, threads, kernel_size, ref, ref_name);
  
  return 0;
}

#endif