#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

__global__ void kernel (int *in)
{
  grid_group grid = this_grid();
  for (int i = 0; i < 10; i++)
  {
    atomicAdd(in, 1);
    grid.sync();
    if (grid.thread_rank() == 0) { printf("%d: %d\n", i, *in); }
    grid.sync();
  }
}

struct arguments {
  int *in;
};

__host__ int main()
{
  int numBlocksPerSm = 0, numThreads = 64;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, kernel, numThreads, 0);
  printf("#threads: %d, #blocks: %d\n", numThreads, numBlocksPerSm);

  struct arguments *args = (struct arguments*) malloc(sizeof(struct arguments));
  int *dev_num = 0, num = 0;
  hipMalloc((void**) &dev_num, sizeof(int));

  hipMemcpy(&dev_num, &num, sizeof(int), hipMemcpyHostToDevice);
  args -> in = dev_num;

  hipLaunchCooperativeKernel(reinterpret_cast<const void*>(kernel), 1 * numBlocksPerSm, numThreads, (void **) &args);
  hipDeviceSynchronize();
  return 0;
}