#include "hip/hip_runtime.h"


#include <dev_dense.cuh>
#include <cuda_timer.cuh>
#include <dense_getrf.cuh>

#define CUDA_DEVICE 0

__global__ void dense_getrf_kernel (double *matrix, const int nx, const int ld, const int ny)
{
  /* 
  * Using 1 block, running parallel both horizontal and vertical 
  */
  const int  thread_id = threadIdx.x, block_size = blockDim.x;

  if (nx * ny > 6 * 1024) /* matrix is too big to load all in shared memory. */
  {
    blockDenseGetrfNoPivot <double> (matrix, nx, ld, ny);
  }
  else /* matrix is small enough to load all in shared memory. */
  {
    extern __shared__ double shm_matrix[];

    for (int i = thread_id; i < nx * ny; i += block_size)
    { 
      const int row = i / nx, col = i - row * nx, index = row * ld + col;
      shm_matrix[i] = matrix[index];
    }
    __syncthreads();
  
    blockDenseGetrfNoPivot <double> (&shm_matrix[0], nx, nx, ny);

    for (int i = thread_id; i < nx * ny; i += block_size)
    { 
      const int row = i / nx, col = i - row * nx, index = row * ld + col;
      matrix[index] = shm_matrix[i];
    }
  }

}

__host__ int dense_getrf_sync (double *matrix, const int nx, const int ld, const int ny) 
{
  if (ld < nx) { printf("GETRF ABORT: Matrix's horizontal offset is less than the number of entries.\n");  return -1; }
  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  dim3 block(256), grid(1);
  hipStream_t main_stream;
  hipStreamCreate(&main_stream);

  const int shm_size = (nx * ny > 6 * 1024) ? 0 : nx * ny * sizeof(double);
  if (shm_size == 0) { printf("WARNING: Matrix size exceeded 48KB of shared memory size. \n-------- Using Global mem instead.\n\n"); }

  struct timer myTimer = timer();
  myTimer.newEvent("GETRF", main_stream);

  //create_timing_event_to_stream ("GETRF TOTAL", main_stream);

  dense_getrf_kernel <<<grid, block, shm_size, main_stream>>> (matrix, nx, ld, ny);

  //create_timing_event_to_stream ("GETRF TOTAL", main_stream);
  myTimer.newEvent("GETRF", main_stream);
  hipStreamDestroy(main_stream);

  myTimer.printStatus();
  myTimer.dumpAllEvents_Sync();

  //device_sync_dump_timed_events ();
  printf("Cuda Execution: getrf finished.\n\n");

  return 0;
}

void test_dense_getrf_nxn (const int nx, const int ny)
{
  printf("-------- Testing %d x %d Dense GETRF: --------\n\n", ny, nx);
  hipSetDevice(CUDA_DEVICE);
  printf("Running on cuda device: %d\n\n", CUDA_DEVICE);

  struct dev_dense <double> *a = new dev_dense <double> (nx, ny);
  a -> loadTestMatrix();
  a -> copyToDevice_Sync();

  dense_getrf_sync(a -> dev_ptr, a -> nx, a -> ld, a -> ny);

  a -> copyToHost_Sync();
  struct dev_dense <double> *b = a -> restoreLU();
  a -> loadTestMatrix();

  printf("Rel. L2 Error: %e\n", b -> L2Error(a));
  printf("-------- n x n Dense GETRF test finished --------\n\n");

  a -> ~dev_dense();
  b -> ~dev_dense();
  free(a);
  free(b);

}
/*
__global__ void dense_trsm_kernel(double *matrix, const  nx, const  ld, const  ny)
{
  
}

__host__ void dense_trsm_sync (Matrix *b, Matrix *a, const double alpha, const bool side, const bool unit_triangular, const bool uplo_lower)
{
  
}

extern "C" void test_inverse (const  nx, const  ny)
{
  hipSetDevice(CUDA_DEVICE);
  printf("Running on cuda device: %d\n", CUDA_DEVICE);

  Matrix a = testMatrix_M(nx, ny);
  dense_getrf_sync(&a);

  Matrix b = identityMatrix_M(ny, ny);
  //dense_trsm_sync (&b, &a, 1, false, false, true, true);

  Matrix c = identityMatrix_M(nx, nx);
  //dense_trsm_sync (&c, &a, 1, true, false, true, true);

  Matrix result0 = matrixMultiplication(testMatrix_M(nx, ny), b);
  Matrix result1 = matrixMultiplication(c, testMatrix_M(nx, ny));

  printf("left inverse: Rel. L2 Error: %e\n", L2Error(result0, identityMatrix_M(ny, ny)));
  printf("right inverse: Rel. L2 Error: %e\n", L2Error(result1, identityMatrix_M(nx, nx)));
}*/


int main(int argc, char **argv)
{
  const int nx = 16;
  const int ny = 16;

  test_dense_getrf_nxn (nx, ny);

  //test_inverse (nx, ny);


  return 0;
}