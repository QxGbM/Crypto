
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <definitions.h>
#include <dense.h>

real_t* Dense::copyToCudaArray() const {
  return copyToCudaArray(nullptr, ld);
}

real_t* Dense::copyToCudaArray(real_t* arr, const int ld_arr) const {
  real_t* e = arr;
  if (arr == nullptr) {
    hipMalloc(&e, (size_t)m * ld_arr * sizeof(real_t));
  }

  if (ld == ld_arr) {
    hipMemcpy(e, elements.data(), (size_t)m * ld * sizeof(real_t), hipMemcpyHostToDevice);
  }
  else {
    for (int y = 0; y < m; y++) {
      hipMemcpy(&e[(size_t)y * ld_arr], &elements.data()[(size_t)y * ld], (size_t)n * sizeof(real_t), hipMemcpyHostToDevice);
    }
  }
  return e;
}

void Dense::copyFromCudaArray(real_t* arr, const int ld_arr) {
  if (ld == ld_arr) {
    hipMemcpy(elements.data(), arr, (size_t)m * ld * sizeof(real_t), hipMemcpyDeviceToHost);
  }
  else {
    for (int y = 0; y < m; y++) {
      hipMemcpy(&elements.data()[(size_t)y * ld], &arr[(size_t)y * ld_arr], (size_t)n * sizeof(real_t), hipMemcpyDeviceToHost);
    }
  }

}