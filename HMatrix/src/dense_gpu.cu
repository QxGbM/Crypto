
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <definitions.h>
#include <dense.h>

real_t* Dense::copyToCudaArray(real_t* arr) const {
  real_t* e = arr;
  if (arr == nullptr) {
    hipMalloc(&e, (size_t) m * n * sizeof(real_t));
  }

  if (ld == n) {
    hipMemcpy(e, elements, (size_t)m * n * sizeof(real_t), hipMemcpyHostToDevice);
  }
  else {
    for (int y = 0; y < m; y++) {
      hipMemcpy(&e[y * n], &elements[y * ld], (size_t)n * sizeof(real_t), hipMemcpyHostToDevice);
    }
  }
  return e;
}
